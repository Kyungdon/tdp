#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <Eigen/Dense>
#include <tdp/hip/hip_runtime.h>
#include <tdp/nvidia/hip/hip_runtime_api.h>
#include <tdp/depth.h>
#include <tdp/image.h>

namespace tdp {

__global__ void KernelDepthConvert(Image<uint16_t> dRaw,
    Image<float> d, 
    float scale, 
    float dMin, 
    float dMax
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if (idx < dRaw.w_ && idy < dRaw.h_) {
    const float di = ((float)dRaw(idx,idy))*scale;
    //if (100<idx&&idx<110 && 100<idy&&idy<110) printf("%f %f %f\n",di,dMin,dMax);
    if (dMin < di && di < dMax) {
      d(idx,idy) = di;
    } else {
      d(idx,idy) = NAN; // nan
    }
  } else if (idx < d.w_ && idy < d.h_) {
    // d might be bigger than dRaw because of consecutive convolutions
    d(idx,idy) = NAN; // nan
  }
}

void ConvertDepth(const Image<uint16_t>& dRaw,
    const Image<float>& d,
    float scale,
    float dMin, 
    float dMax
    ) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,d,32,32);
  //std::cout << blocks.x << " " << blocks.y << " " << blocks.z << std::endl;
  KernelDepthConvert<<<blocks,threads>>>(dRaw,d,scale,dMin,dMax);
  checkCudaErrors(hipDeviceSynchronize());
}

}
