#include "hip/hip_runtime.h"
/* Copyright (c) 2016, Julian Straub <jstraub@csail.mit.edu> Licensed
 * under the MIT license. See the license file LICENSE.
 */

#include <assert.h>
#include <tdp/eigen/dense.h>
#include <tdp/hip/hip_runtime.h>
#include <tdp/nvidia/hip/hip_runtime_api.h>
#include <tdp/image.h>
#include <tdp/managed_image.h>
#include <tdp/camera.h>
#include <tdp/reductions.cuh>
#include <tdp/manifold/SE3.h>

namespace tdp {

// R_mc: R_model_current
template<int BLK_SIZE>
__global__ void KernelICPStep(
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_c,
    Image<Vector3fda> n_c,
    Matrix3fda R_mc, 
    Vector3fda t_mc, 
    const Camera<float> cam,
    float dotThr,
    float distThr,
    int N_PER_T,
    Image<float> out
    ) {
  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = idx*N_PER_T;
  const int N = pc_m.w_*pc_m.h_;
  const int idE = min(N,(idx+1)*N_PER_T);
  __shared__ Eigen::Matrix<float,29,1,Eigen::DontAlign> sum[BLK_SIZE];
  sum[tid] = Eigen::Matrix<float,29,1,Eigen::DontAlign>::Zero();
  for (int id=idS; id<idE; ++id) {
    const int idx = id%pc_c.w_;
    const int idy = id/pc_c.w_;
    Vector3fda pc_ci = pc_c(idx,idy);
    // project current point into model frame to get association
    if (idx < pc_c.w_ && idy < pc_c.h_ && IsValidData(pc_ci)) {
      Vector3fda pc_c_in_m = R_mc * pc_ci + t_mc;
      // project into model camera
      Vector2fda x_c_in_m = cam.Project(pc_c_in_m);
      int u = floor(x_c_in_m(0)+0.5f);
      int v = floor(x_c_in_m(1)+0.5f);
      if (0 <= u && u < pc_m.w_ && 0 <= v && v < pc_m.h_
          && pc_ci(2) > 0. && pc_c_in_m(2) > 0.
          && IsValidData(pc_c_in_m)) {
        // found association -> check thresholds;
        Vector3fda n_c_in_m = R_mc * n_c(idx,idy);
        Vector3fda n_mi = n_m(u,v);
        Vector3fda pc_mi = pc_m(u,v);
        float dot  = n_mi.dot(n_c_in_m);
        float dist = (pc_mi-pc_c_in_m).norm();
        //if (tid < 10)
        //  printf("%d %d to %d %d; 3d: %f %f %f; %f >? %f\n",idx,idy,u,v,pc_c(idx,idy)(0),pc_c(idx,idy)(1),pc_c(idx,idy)(2),dot,dotThr);
        if (dot > dotThr && dist < distThr && IsValidData(pc_mi)) {
          // association is good -> accumulate
          // if we found a valid association accumulate the A and b for A x = b
          // where x \in se{3} as well as the residual error
          float ab[7];      
          Eigen::Map<Vector3fda> top(&(ab[0]));
          Eigen::Map<Vector3fda> bottom(&(ab[3]));
          // lowkl
          //top = (pc_c_in_m).cross(n_mi);
          // as in Kinfu paper: 
          //top = (n_mi).cross(pc_c_in_m);
          // as in my own deriv: top = (R_mc * pc_ci).cross(n_mi);
          // as in mp3guy: 
          top = (pc_c_in_m).cross(n_mi);
          bottom = n_mi;
          ab[6] = n_mi.dot(pc_mi-pc_c_in_m);
          // lowkl
          //ab[6] = n_mi.dot(-pc_mi+pc_c_in_m);
          assert(ab[3]==n_mi(0));
          assert(ab[4]==n_mi(1));
          assert(ab[5]==n_mi(2));
          Eigen::Matrix<float,29,1,Eigen::DontAlign> upperTriangle;
          int k=0;
#pragma unroll
          for (int i=0; i<7; ++i) {
            for (int j=i; j<7; ++j) {
              upperTriangle(k++) = ab[i]*ab[j];
            }
          }
          assert(k==28);
          upperTriangle(28) = 1.; // to get number of data points
          sum[tid] += upperTriangle;
        }
      }
    }
  }
  __syncthreads(); //sync the threads
#pragma unroll
  for(int s=(BLK_SIZE)/2; s>1; s>>=1) {
    if(tid < s) {
      sum[tid] += sum[tid+s];
    }
    __syncthreads();
  }
  if(tid < 29) {
    // sum the last two remaining matrixes directly into global memory
    atomicAdd(&out[tid], sum[0](tid)+sum[1](tid));
    //atomicAdd_<float>();
    //printf("%f %f %f \n",out[tid],sum[0](tid),sum[1](tid));
  }
}

void ICPStep (
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_c,
    Image<Vector3fda> n_c,
    Matrix3fda& R_mc, 
    Vector3fda& t_mc, 
    const Camera<float>& cam,
    float dotThr,
    float distThr,
    Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb,
    float& error,
    float& count
    ) {
  size_t N = pc_m.w_*pc_m.h_;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,N/10,256);
  ManagedDeviceImage<float> out(29,1);
  hipMemset(out.ptr_, 0, 29*sizeof(float));

  KernelICPStep<256><<<blocks,threads>>>(pc_m,n_m,pc_c,n_c,R_mc,t_mc,cam,
      dotThr,distThr,10,out);
  checkCudaErrors(hipDeviceSynchronize());
  ManagedHostImage<float> sumAb(29,1);
  hipMemcpy(sumAb.ptr_,out.ptr_,29*sizeof(float), hipMemcpyDeviceToHost);

  //for (int i=0; i<29; ++i) std::cout << sumAb[i] << "\t";
  //std::cout << std::endl;
  ATA.fill(0.);
  ATb.fill(0.);
  int k = 0;
  for (int i=0; i<6; ++i) {
    for (int j=i; j<7; ++j) {
      float val = sumAb[k++];
      if (j==6)  {
        ATb(i) = val;
      } else {
        ATA(i,j) = val;
        ATA(j,i) = val;
      }
    }
  }
  count = sumAb[28];
  error = sumAb[27]/count;
  //std::cout << ATA << std::endl << ATb.transpose() << std::endl;
  //std::cout << "\terror&count " << error << " " << count << std::endl;
}

// R_mc: R_model_current
template<int BLK_SIZE>
__global__ void KernelICPVisualizeAssoc(
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_c,
    Image<Vector3fda> n_c,
    Matrix3fda R_mc, 
    Vector3fda t_mc, 
    const Camera<float> cam,
    float dotThr,
    float distThr,
    int N_PER_T,
    Image<float> assoc_m,
    Image<float> assoc_c
    ) {
  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = idx*N_PER_T;
  const int N = pc_m.w_*pc_m.h_;
  const int idE = min(N,(idx+1)*N_PER_T);

  for (int id=idS; id<idE; ++id) {
    const int idx = id%pc_c.w_;
    const int idy = id/pc_c.w_;
    Vector3fda pc_ci = pc_c(idx,idy);
    // project current point into model frame to get association
    if (idx < pc_c.w_ && idy < pc_c.h_ && IsValidData(pc_ci)) {
      Vector3fda pc_c_in_m = R_mc * pc_ci + t_mc;
      // project into model camera
      Vector2fda x_c_in_m = cam.Project(pc_c_in_m);
      int u = floor(x_c_in_m(0)+0.5f);
      int v = floor(x_c_in_m(1)+0.5f);
      if (0 <= u && u < pc_m.w_ && 0 <= v && v < pc_m.h_
          && pc_ci(2) > 0. && pc_c_in_m(2) > 0.
          && IsValidData(pc_c_in_m)) {
        // found association -> check thresholds;
        Vector3fda n_c_in_m = R_mc * n_c(idx,idy);
        Vector3fda n_mi = n_m(u,v);
        Vector3fda pc_mi = pc_m(u,v);
        float dot  = n_mi.dot(n_c_in_m);
        float dist = (pc_mi-pc_c_in_m).norm();
        if (dot > dotThr && dist < distThr && IsValidData(pc_mi)) {
          // association is good -> accumulate
          //assoc_m(u,v) = n_mi.dot(-pc_mi+pc_c_in_m);
          //assoc_c(idx,idy) = n_mi.dot(-pc_mi+pc_c_in_m);
          assoc_m(u,v) = (-pc_mi+pc_c_in_m).norm();
          assoc_c(idx,idy) = (-pc_mi+pc_c_in_m).norm();
        }
      }
    }
  }
}

void ICPVisualizeAssoc (
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_c,
    Image<Vector3fda> n_c,
    SE3f& T_mc,
    const Camera<float>& cam,
    float angleThr,
    float distThr,
    Image<float>& assoc_m,
    Image<float>& assoc_c
    ) {
  size_t N = pc_m.w_*pc_m.h_;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,N/10,256);
  Matrix3fda R_mc = T_mc.rotation().matrix();
  Vector3fda t_mc = T_mc.translation();
  KernelICPVisualizeAssoc<256><<<blocks,threads>>>(pc_m,n_m,pc_c,n_c,
      R_mc,t_mc,cam,
      cos(angleThr*M_PI/180.),distThr,10,assoc_m, assoc_c);
  checkCudaErrors(hipDeviceSynchronize());
}

}
