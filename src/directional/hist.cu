#include "hip/hip_runtime.h"

#include <Eigen/Dense>
#include <tdp/image.h>
#include <tdp/hip/hip_runtime.h>
#include <tdp/nvidia/hip/hip_runtime_api.h>

namespace tdp {

__global__
void KernelComputeCentroidBasedGeodesicHist(
    Image<Eigen::Vector3f> n,
    Image<Eigen::Vector3f> tri_centers,
    Image<uint32_t> hist
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if (idx < n.w_ && idy < n.h_) {
    int id = 0;
    Eigen::Vector3f ni = n(idx,idy);
    float maxDot = -1;
    for (uint32_t i=0; i<tri_centers.w_; ++i) {
      float dot = ni.dot(tri_centers[i]);
      if (dot > maxDot) {
        maxDot = dot;
        id =i;
      }
    }
    atomicInc(&hist[id], 2147483647);
  }
}

void ComputeCentroidBasedGeoidesicHist(
    Image<Eigen::Vector3f>& n,
    Image<Eigen::Vector3f>& tri_centers,
    Image<uint32_t>& hist
    ) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,n,32,32);
  KernelComputeCentroidBasedGeodesicHist<<<blocks,threads>>>(n,tri_centers,hist);
  checkCudaErrors(hipDeviceSynchronize());
}

}
