#include "hip/hip_runtime.h"

#include <tdp/eigen/dense.h>
#include <iostream>
#include <tdp/data/image.h>
#include <tdp/hip/hip_runtime.h>
#include <tdp/nvidia/hip/hip_runtime_api.h>

namespace tdp {

__global__
void KernelComputeCentroidBasedGeodesicHist(
    Image<tdp::Vector3fda> n,
    Image<tdp::Vector3fda> tri_centers,
    Image<uint32_t> hist
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if (idx < n.w_ && idy < n.h_) {
    tdp::Vector3fda ni = n(idx,idy);
    if (isnan(ni(0)) || isnan(ni(1)) || isnan(ni(2)) 
        || ni.norm() < 0.9) return;
    int id = 0;
    int N = tri_centers.w_;
    float maxDot = -1;
    for (uint32_t i=0; i<N; ++i) {
      const tdp::Vector3fda& c = tri_centers[i];
      float dot = ni.dot(c);
      if (dot > maxDot) {
        maxDot = dot;
        id =i;
      }
    }
    atomicInc(&hist[id], 2147483647);
  }
}

void ComputeCentroidBasedGeoidesicHist(
    Image<tdp::Vector3fda>& n,
    Image<tdp::Vector3fda>& tri_centers,
    Image<uint32_t>& hist
    ) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,n,32,32);
  KernelComputeCentroidBasedGeodesicHist<<<blocks,threads>>>(n,tri_centers,hist);
  checkCudaErrors(hipDeviceSynchronize());
}

}
