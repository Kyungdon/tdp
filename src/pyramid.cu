#include "hip/hip_runtime.h"
/* Copyright (c) 2016, Julian Straub <jstraub@csail.mit.edu> Licensed
 * under the MIT license. See the license file LICENSE.
 */

#include <assert.h>
#include <tdp/pyramid.h>
#include <tdp/image.h>
#include <tdp/hip/hip_runtime.h>
#include <tdp/nvidia/hip/hip_runtime_api.h>

namespace tdp {

template<typename T>
__global__
void KernelPyrDown(
    const Image<T> Iin,
    Image<T> Iout
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if (idx < Iout.w_ && idy < Iout.h_) {
    T* in0 = Iin.RowPtr(idy*2);
    T* in1 = Iin.RowPtr(idy*2+1);
    Iout(idx,idy) = 0.25f*(in0[idx*2] + in0[idx*2+1] + in1[idx*2] + in1[idx*2+1]);
  }
}

void PyrDown(
    const Image<float>& Iin,
    Image<float>& Iout
    ) {
  //printf("%dx%d %dx%d\n",Iin.w_,Iin.h_,Iout.w_,Iout.h_);
  assert(Iin.w_ == Iout.w_*2);
  assert(Iin.h_ == Iout.h_*2);
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,Iout,32,32);
  KernelPyrDown<float><<<blocks,threads>>>(Iin,Iout);
  checkCudaErrors(hipDeviceSynchronize());
}
}
