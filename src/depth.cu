#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <Eigen/Dense>
#include <tdp/depth.h>
#include <tdp/image.h>

namespace tdp {

__global__ void KernelDepthConvert(Image<uint16_t> dRaw,
    Image<float> d, float scale) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if (idx < dRaw.w && idy < dRaw.h) {
    const uint16_t di = RowPtr<uint16_t>(dRaw,idy)[idx];
    if (di > 0) {
      RowPtr<float>(d,idy)[idx] = ((float)di)*scale;
    } else {
      RowPtr<float>(d,idy)[idx] = 0./0.; // nan
    }
  } else if (idx < d.w && idy < d.h) {
    // d might be bigger than dRaw because of consecutive convolutions
    RowPtr<float>(d,idy)[idx] = 0./0.; // nan
  }
}

void ConvertDepth(const Image<uint16_t>& dRaw,
    const Image<float>& d,
    float scale) {
  size_t w = d.w;
  size_t h = d.h;
  dim3 threads(32,32,1);
  dim3 blocks(w/32+(w%32>0?1:0), h/32+(h%32>0?1:0),1);
  KernelDepthConvert<<<blocks,threads>>>(dRaw,d,scale);

}

}
