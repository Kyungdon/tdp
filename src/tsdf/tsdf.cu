#include "hip/hip_runtime.h"

#include <math.h>
#include <iostream>
#include <Eigen/Core>
#include <Eigen/Dense>
#include <tdp/cuda/hip/hip_runtime.h>
#include <tdp/tsdf/tsdf.h>
#include <tdp/camera/projective_math.h>
#include <tdp/camera/camera_poly.h>
#include <tdp/camera/camera.h>
#include <tdp/camera/ray.h>

namespace tdp {

__device__
inline Vector3fda NormalFromTSDF(int x, int y, int z, float tsdfVal,
    const Volume<TSDFval>& tsdf, const Vector3fda& dGrid) {
  // surface normal: TODO might want to do better interpolation
  // of neighbors
  Vector3fda ni (
      (x+1 < tsdf.w_)? tsdf(x+1,y,z).f - tsdfVal
      : tsdfVal - tsdf(x-1,y,z).f,
      (y+1 < tsdf.h_)? tsdf(x,y+1,z).f - tsdfVal
      : tsdfVal - tsdf(x,y-1,z).f,
      (z+1 < tsdf.d_)? tsdf(x,y,z+1).f - tsdfVal
      : tsdfVal - tsdf(x,y,z-1).f);
  // apply weighting according to TSDF volume voxel side length
  // TODO: this is still not working: only nice results with d_==w_==h_
  ni(0) /= dGrid(0);
  ni(1) /= dGrid(1);
  ni(2) /= dGrid(2);
  // negate to flip the normals to face the camera
  return -ni/ni.norm();
}

__device__
inline bool RayTraceTSDFinZonly(
    const Rayfda& r_d_in_r,
    const Vector3fda& grid0,
    const Vector3fda& dGrid,
    const Volume<TSDFval>& tsdf,
    float& d,
    Vector3ida& idTSDF
    ) {
  // iterate over z in TSDF; detect 0 crossing in TSDF
  float tsdfValPrev = -1.01;
  float di_Prev = 0.;
  for (size_t idz=0; idz<tsdf.d_; ++idz) {
    float z = grid0(2)+idz*dGrid(2);  // depth
    // intersect r_d_in_r with plane at depth z in TSDF coordinates
    // to get depth along r_d_in_r
    //float d = (-z - T_rd.translation().dot(n))/(r_r.dot(n));
    // since n is (0,0,-1):
    float di = (-z+r_d_in_r.p(2))/(-r_d_in_r.dir(2));
    if (di < 0.) continue; // ignore things behind
    // get intersection point in TSDF volume at depth z
    Vector3fda u_r = r_d_in_r.PointAtDepth(di);
    int x = floor((u_r(0)-grid0(0))/dGrid(0)+0.5);
    int y = floor((u_r(1)-grid0(1))/dGrid(1)+0.5);
    if (0<=x&&x<tsdf.w_ && 0<=y&&y<tsdf.h_) {
      float tsdfVal = tsdf(x,y,idz).f;
      float tsdfW = tsdf(x,y,idz).w;
      if (tsdfW > 5 && -1 < tsdfVal && tsdfVal <= 0. && tsdfValPrev >= 0.) {
        // detected 0 crossing -> interpolate
        d = di_Prev -((di-di_Prev)*tsdfValPrev)/(tsdfVal-tsdfValPrev);
        idTSDF(0) = x;
        idTSDF(1) = y;
        idTSDF(2) = idz;
        return true;
      }
      tsdfValPrev = tsdfVal;
    }
    di_Prev = di;
  }
  return false;
}

__device__
inline bool RayTraceTSDF(
    const Rayfda& r_d_in_r,
    const Vector3fda& grid0,
    const Vector3fda& dGrid,
    const Volume<TSDFval>& tsdf,
    float& d,
    Vector3ida& idTSDF,
    float mu,
    float wThr,
    bool verbose = false
    ) {
  // iterate over z in TSDF; detect 0 crossing in TSDF
  float tsdfValPrev = -1.01;
  float di_Prev = 0.;
  // find the dimension of TSDF Volume that is best aligned with the
  // ray direction
  int dimIt = 0;
  r_d_in_r.dir.array().abs().maxCoeff(&dimIt);
  int idItMax = dimIt == 0 ? tsdf.w_ : (dimIt == 1 ? tsdf.h_ : tsdf.d_);
  int idItMin = 0;
  int dimInc = r_d_in_r.dir(dimIt) < 0 ? -1 : 1;
  if (dimInc < 0) {
    idItMin = idItMax - 1;
    idItMax = -1;
  }

  // start from where the camera is currently located in the TSDF
  idItMin = (r_d_in_r.p(dimIt)-grid0(dimIt))/dGrid(dimIt);

//  if (verbose) {
//    printf("%f %f %f: %d %d, (%d %d)\n",  r_d_in_r.dir(0),  r_d_in_r.dir(1),
//      r_d_in_r.dir(2), dimIt, dimInc, idItMin, idItMax);
//  }

//  for (int idIt = idItMin; idIt != idItMax; idIt += dimInc) {
  int idIt = idItMin;
  while(idItMax - idIt*dimInc > 0) {
    Vector3fda nOverD = Vector3fda::Zero();
    //nOverD(dimIt) = -dimInc/(grid0(dimIt)+idIt*dGrid(dimIt));
    nOverD(dimIt) = -1./(grid0(dimIt)+idIt*dGrid(dimIt));
    // to get depth along r_d_in_r
    float di = (-1 - r_d_in_r.p.dot(nOverD))/(r_d_in_r.dir.dot(nOverD));
    if (di > 0.) { // only of surfaces is in front of camera
      // get intersection point in TSDF volume at depth z
      Vector3fda u_r = r_d_in_r.PointAtDepth(di);
      int x = floor((u_r(0)-grid0(0))/dGrid(0)+0.5);
      int y = floor((u_r(1)-grid0(1))/dGrid(1)+0.5);
      int z = floor((u_r(2)-grid0(2))/dGrid(2)+0.5);
      if (0<=x&&x<tsdf.w_ && 0<=y&&y<tsdf.h_ && 0<=z&&z<tsdf.d_) {
        float tsdfVal = tsdf(x,y,z).f;
        float tsdfW = tsdf(x,y,z).w;
        if (tsdfW > wThr && -1 < tsdfVal
            && tsdfVal <= 0. && tsdfValPrev >= 0.) {
          // detected 0 crossing -> interpolate
          d = di_Prev-((di-di_Prev)*tsdfValPrev)/(tsdfVal-tsdfValPrev);
          idTSDF(0) = x;
          idTSDF(1) = y;
          idTSDF(2) = z;
          return true;
        }
        tsdfValPrev = tsdfVal;
        if (tsdfVal >= 1.f) {
          // if we are still far from the surface take larger steps
          idIt += dimInc*(floor(mu/dGrid(dimIt))-1);
        }
      }
      di_Prev = di;
    }
    idIt += dimInc;
  }
  return false;
}

// ray trace and compute depth image as well as normals from pose T_rd
template<int D, typename Derived>
__global__
void KernelRayTraceTSDF(Volume<TSDFval> tsdf, Image<float> d,
    Image<Vector3fda> n, SE3f T_rd,
    CameraBase<float,D,Derived> camD,
    Vector3fda grid0, Vector3fda dGrid, float mu, float wThr) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if (idx < d.w_ && idy < d.h_) {
    d(idx,idy) = NAN;
    n(idx,idy)(0) = NAN;
    n(idx,idy)(1) = NAN;
    n(idx,idy)(2) = NAN;
    // ray of depth image d
    Rayfda r_d(Vector3fda::Zero(), camD.Unproject(idx, idy, 1.));
    // ray of depth image d in reference coordinates (TSDF)
    Rayfda r_d_in_r = r_d.Transform(T_rd);

    float di = 0;
    Vector3ida idTSDF;
    if (RayTraceTSDF(r_d_in_r, grid0, dGrid, tsdf, di, idTSDF, mu, wThr)) {

//          idx==d.w_/2 && idy==d.h_/2)) {
      // depth
      d(idx,idy) = di;
      // surface normal:
      Vector3fda ni = NormalFromTSDF(idTSDF(0),idTSDF(1),idTSDF(2),
          tsdf(idTSDF(0),idTSDF(1),idTSDF(2)).f, tsdf, dGrid);
      // and compute the normal in the depth frame of reference
      n(idx,idy) = T_rd.rotation().InverseTransform(ni);
    }
  }
}


template<int D, typename Derived>
__global__
void KernelAddToTSDF(Volume<TSDFval> tsdf, Image<float> d,
    SE3f T_rd, SE3f T_dr, CameraBase<float,D,Derived>camD,
    Vector3fda grid0, Vector3fda dGrid, float mu, float wMax) {
  // kernel over all pixel locations and depth locations in the TSDF
  // volume
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  const int idz = threadIdx.z + blockDim.z * blockIdx.z;

  if (idx < tsdf.w_ && idy < tsdf.h_ && idz < tsdf.d_) {
    // unproject point in reference frame
    Eigen::Vector3f p_r (grid0(0) + idx*dGrid(0),
        grid0(1)+idy*dGrid(1),
        grid0(2)+idz*dGrid(2));
    // project the point into the depth frame
    Eigen::Vector3f p_d = T_dr*p_r;
    if (p_d(2) < 0.) return; // dont add to behind the camera.
    Eigen::Vector2f u_d = camD.Project(p_d);
    int x = floor(u_d(0)+0.5);
    int y = floor(u_d(1)+0.5);
    if (0<=x&&x<d.w_ && 0<=y&&y<d.h_) {
      const float z_d = d(x, y);
      if (z_d > 0.1) {
        const float lambda = camD.Unproject(u_d(0),u_d(1),1.).norm();
        const float z_tsdf = (T_rd.translation()-p_r).norm()/lambda;
        const float eta = z_d - z_tsdf;
        if (eta >= -mu) {
          const float etaOverMu = eta/mu;
          const float psi = (etaOverMu>1.f?1.f:etaOverMu);
          // TODO can use other weights as well (like incidence angle)
          const float Wnew = 1.;
          const float Wprev = tsdf(idx,idy,idz).w;
          tsdf(idx,idy,idz).f = (Wprev*tsdf(idx,idy,idz).f
              + Wnew*psi)/(Wprev+Wnew);
          tsdf(idx,idy,idz).w = min(Wprev+Wnew, wMax);
        }
      }
    }
  }
}

template<int D, typename Derived>
__global__
void KernelAddToTSDF(Volume<TSDFval> tsdf, Image<float> d, Image<Vector3bda> rgb,
    SE3f T_rd, SE3f T_dr, CameraBase<float,D,Derived>camD,
    Vector3fda grid0, Vector3fda dGrid, float mu) {
  // kernel over all pixel locations and depth locations in the TSDF
  // volume
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  const int idz = threadIdx.z + blockDim.z * blockIdx.z;

  if (idx < tsdf.w_ && idy < tsdf.h_ && idz < tsdf.d_) {
    // unproject point in reference frame
    Eigen::Vector4f p_r (grid0(0) + idx*dGrid(0),
        grid0(1)+idy*dGrid(1),
        grid0(2)+idz*dGrid(2),1);

    // project the point into the depth frame
    Eigen::Vector3f p_d = T_dr.matrix3x4()*p_r;
    if (p_d(2) < 0.) return; // dont add to behind the camera.
    Eigen::Vector2f u_d = camD.Project(p_d);
    int x = floor(u_d(0)+0.5);
    int y = floor(u_d(1)+0.5);

    if (0 <= x && x < d.w_ && 0 <= y && y < d.h_) {
      const float z_d = d(x, y);
      const float lambda = camD.Unproject(u_d(0),u_d(1),1.).norm();
      const float z_tsdf = (T_rd.translation()-p_r.topRows<3>()).norm()/lambda;
      const float eta = z_d - z_tsdf;
      if (eta >= -mu) {
        const float etaOverMu = eta/mu;
        const float psi = (etaOverMu>1.f?1.f:etaOverMu);
        // TODO can use other weights as well (like incidence angle)
        const float Wnew = 1.;
        const float Wprev = tsdf(idx,idy,idz).w;

        tsdf(idx, idy, idz).f = (Wprev*tsdf(idx,idy,idz).f
            + Wnew*psi)/(Wprev+Wnew);
        tsdf(idx, idy, idz).w = min(Wprev + Wnew, 100.f);

        // tsdf(idx, idy, idz).rgb = rgb(x, y);
        tsdf(idx, idy, idz).r = rgb(x, y)(0);
        tsdf(idx, idy, idz).g = rgb(x, y)(1);
        tsdf(idx, idy, idz).b = rgb(x, y)(2);
      }
    }
  }
}

template<int D, typename Derived>
void TSDF::AddToTSDF(Volume<TSDFval> tsdf, Image<float> d,
    SE3f T_rd, CameraBase<float,D,Derived>camD,
    Vector3fda grid0, Vector3fda dGrid,
    float mu, float wMax) {
  dim3 threads, blocks;
  ComputeKernelParamsForVolume(blocks,threads,tsdf,8,8,8);
  KernelAddToTSDF<<<blocks,threads>>>(tsdf, d, T_rd, T_rd.Inverse(),
      camD, grid0, dGrid, mu, wMax);
  checkCudaErrors(hipDeviceSynchronize());
}

template void TSDF::AddToTSDF(Volume<TSDFval> tsdf, Image<float> d,
    SE3f T_rd,
    CameraBase<float,Camera<float>::NumParams,Camera<float>> camD,
    Vector3fda grid0, Vector3fda dGrid,
    float mu, float wMax);
template void TSDF::AddToTSDF(Volume<TSDFval> tsdf, Image<float> d,
    SE3f T_rd,
    CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>> camD,
    Vector3fda grid0, Vector3fda dGrid,
    float mu, float wMax);

template<int D, typename Derived>
void TSDF::AddToTSDF(Volume<TSDFval> tsdf, Image<float> d, Image<Vector3bda> rgb,
    SE3f T_rd, CameraBase<float,D,Derived>camD,
    Vector3fda grid0, Vector3fda dGrid,
    float mu) {
  dim3 threads, blocks;
  ComputeKernelParamsForVolume(blocks, threads, tsdf, 8, 8, 8);

  KernelAddToTSDF<<<blocks,threads>>>(tsdf, d, rgb, T_rd, T_rd.Inverse(),
      camD, grid0, dGrid, mu);

  checkCudaErrors(hipDeviceSynchronize());
}

template void TSDF::AddToTSDF(Volume<TSDFval> tsdf, Image<float> d, Image<Vector3bda> rgb,
    SE3f T_rd,
    CameraBase<float,Camera<float>::NumParams,Camera<float>> camD,
    Vector3fda grid0, Vector3fda dGrid,
    float mu);
template void TSDF::AddToTSDF(Volume<TSDFval> tsdf, Image<float> d, Image<Vector3bda> rgb,
    SE3f T_rd,
    CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>> camD,
    Vector3fda grid0, Vector3fda dGrid,
    float mu);

template<int D, typename Derived>
void TSDF::RayTraceTSDF(Volume<TSDFval> tsdf, Image<float> d, Image<Vector3fda> n,
    SE3f T_rd,
    CameraBase<float,D,Derived> camD,
    Vector3fda grid0, Vector3fda dGrid,
    float mu, float wThr) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,d,32,32);
  KernelRayTraceTSDF<D,Derived><<<blocks,threads>>>(tsdf, d, n, T_rd, camD,
      grid0, dGrid, mu, wThr);
  checkCudaErrors(hipDeviceSynchronize());
}

template void TSDF::RayTraceTSDF(Volume<TSDFval> tsdf, Image<float> d,
    Image<Vector3fda> n, SE3f T_rd,
    CameraBase<float,Camera<float>::NumParams,Camera<float>> camD,
    Vector3fda grid0, Vector3fda dGrid, float mu, float wThr);
template void TSDF::RayTraceTSDF(Volume<TSDFval> tsdf, Image<float> d,
    Image<Vector3fda> n, SE3f T_rd,
    CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>> camD,
    Vector3fda grid0, Vector3fda dGrid, float mu, float wThr);

// T_rd is transformation from depth/camera cosy to reference/TSDF cosy
template<int D, typename Derived>
__global__
void KernelRayTraceTSDF(Volume<TSDFval> tsdf,
    Image<Vector3fda> pc_d,
    Image<Vector3fda> n_d,
    SE3f T_rd, CameraBase<float,D,Derived> camD,
    Vector3fda grid0, Vector3fda dGrid, float mu, float wThr) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if (idx < pc_d.w_ && idy < pc_d.h_) {
    pc_d(idx,idy)(0) = NAN;
    pc_d(idx,idy)(1) = NAN;
    pc_d(idx,idy)(2) = NAN;
    n_d(idx,idy)(0) = NAN;
    n_d(idx,idy)(1) = NAN;
    n_d(idx,idy)(2) = NAN;

    // ray of depth image d
    Rayfda r_d(Vector3fda::Zero(), camD.Unproject(idx, idy, 1.));
    // ray of depth image d in reference coordinates (TSDF)
    Rayfda r_d_in_r = r_d.Transform(T_rd);

    float di = 0;
    Vector3ida idTSDF;
    if (RayTraceTSDF(r_d_in_r, grid0, dGrid, tsdf, di, idTSDF, mu, wThr
//          (idx==pc_d.w_-1 && idy==pc_d.h_-1) ||
//          (idx==0 && idy==pc_d.h_-1) ||
//          (idx==pc_d.w_-1 && idy==0) ||
//          (idx==0 && idy==0)
          )) {
      // surface normal:
      Vector3fda ni = NormalFromTSDF(idTSDF(0),idTSDF(1),idTSDF(2),
          tsdf(idTSDF(0),idTSDF(1),idTSDF(2)).f, tsdf, dGrid);
      // and compute the normal in the depth frame of reference
//      n_d(idx,idy) = T_rd.rotation().Inverse() * ni;
//      pc_d(idx,idy) = r_d.dir*di;
      // compute pc and normals in reference coordinates
      n_d(idx,idy) = ni;
      pc_d(idx,idy) = r_d_in_r.PointAtDepth(di);
    }
  }
}

template<int D, typename Derived>
void TSDF::RayTraceTSDF(Volume<TSDFval> tsdf,
    Image<Vector3fda> pc_d,
    Image<Vector3fda> n_d,
    SE3f T_rd, CameraBase<float,D,Derived>camD,
    Vector3fda grid0, Vector3fda dGrid,
    float mu, float wThr) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,pc_d,32,32);
  KernelRayTraceTSDF<<<blocks,threads>>>(tsdf, pc_d, n_d, T_rd, camD,
      grid0, dGrid, mu, wThr);
  checkCudaErrors(hipDeviceSynchronize());
}

// explicit instantiations
template void TSDF::RayTraceTSDF(Volume<TSDFval> tsdf,
    Image<Vector3fda> pc_d,
    Image<Vector3fda> n_d,
    SE3f T_rd,
    CameraBase<float,Camera<float>::NumParams,Camera<float>> camD,
    Vector3fda grid0, Vector3fda dGrid,
    float mu, float wThr);
template void TSDF::RayTraceTSDF(Volume<TSDFval> tsdf,
    Image<Vector3fda> pc_d,
    Image<Vector3fda> n_d,
    SE3f T_rd,
    CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>> camD,
    Vector3fda grid0, Vector3fda dGrid,
    float mu, float wThr);

}
