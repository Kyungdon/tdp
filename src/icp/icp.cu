#include "hip/hip_runtime.h"
/* Copyright (c) 2016, Julian Straub <jstraub@csail.mit.edu> Licensed
 * under the MIT license. See the license file LICENSE.
 */

#include <assert.h>
#include <tdp/eigen/dense.h>
#include <tdp/cuda/hip/hip_runtime.h>
#include <tdp/nvidia/hip/hip_runtime_api.h>
#include <tdp/data/image.h>
#include <tdp/data/managed_image.h>
#include <tdp/camera/camera.h>
#include <tdp/camera/camera_poly.h>
#include <tdp/reductions/reductions.cuh>
#include <tdp/manifold/SE3.h>
#include <tdp/cuda/cuda.cuh>

namespace tdp {

// T_mc: R_model_observation
template<int BLK_SIZE, int D, typename Derived>
__global__ void KernelICPStep(
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    SE3f T_mo, 
    SE3f T_cm, 
    const CameraBase<float,D,Derived> cam,
    float dotThr,
    float distThr,
    int N_PER_T,
    Image<float> out
    ) {
  assert(BLK_SIZE >=29);
  const int tid = threadIdx.x;
  const int id_ = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = id_*N_PER_T;
  const int N = pc_m.w_*pc_m.h_;
  const int idE = min(N,(id_+1)*N_PER_T);
  SharedMemory<Vector29fda> smem;
  Vector29fda* sum = smem.getPointer();
  sum[tid] = Vector29fda::Zero();
  for (int id=idS; id<idE; ++id) {
    const int idx = id%pc_o.w_;
    const int idy = id/pc_o.w_;
    // project current point into model frame to get association
    if (idx < pc_o.w_ && idy < pc_o.h_ ) {
      Vector3fda pc_oi = pc_o(idx,idy);
      if (!IsValidData(pc_oi)) continue;
      Vector3fda pc_o_in_m = T_mo*pc_oi;
      // project into model camera
      Vector2fda x_o_in_mc = cam.Project(T_cm*pc_o_in_m);
      const int u = floor(x_o_in_mc(0)+0.5f);
      const int v = floor(x_o_in_mc(1)+0.5f);
      if (0 <= u && u < pc_m.w_ && 0 <= v && v < pc_m.h_
          && pc_oi(2) > 0. && pc_o_in_m(2) > 0.
          && IsValidData(pc_o_in_m)) {
        // found association -> check thresholds;
        Vector3fda n_o_in_m = T_mo.rotation()*n_o(idx,idy);
        Vector3fda n_mi = n_m(u,v);
        Vector3fda pc_mi = pc_m(u,v);
        const float dot  = n_mi.dot(n_o_in_m);
        const float dist = (pc_mi-pc_o_in_m).norm();
        if (dot > dotThr && dist < distThr && IsValidData(pc_mi)) {
          // association is good -> accumulate
          // if we found a valid association accumulate the A and b for A x = b
          // where x \in se{3} as well as the residual error
          float ab[7];      
          Eigen::Map<Vector3fda> top(&(ab[0]));
          Eigen::Map<Vector3fda> bottom(&(ab[3]));
          // as in mp3guy: 
          top = (pc_o_in_m).cross(n_mi);
          bottom = n_mi;
          ab[6] = n_mi.dot(pc_mi-pc_o_in_m);
          Eigen::Matrix<float,29,1,Eigen::DontAlign> upperTriangle;
          int k=0;
#pragma unroll
          for (int i=0; i<7; ++i) {
            for (int j=i; j<7; ++j) {
              upperTriangle(k++) = ab[i]*ab[j];
            }
          }
          upperTriangle(28) = 1.; // to get number of data points
          sum[tid] += upperTriangle;
        }
      }
    }
  }
  __syncthreads(); //sync the threads
#pragma unroll
  for(int s=(BLK_SIZE)/2; s>1; s>>=1) {
    if(tid < s) {
      sum[tid] += sum[tid+s];
    }
    __syncthreads();
  }
  if(tid < 29) {
    // sum the last two remaining matrixes directly into global memory
    atomicAdd(&out[tid], sum[0](tid)+sum[1](tid));
  }
}

template<int D, typename Derived>
void ICPStep (
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    const SE3f& T_mo, 
    const SE3f& T_cm,
    const CameraBase<float,D,Derived>& cam,
    float dotThr,
    float distThr,
    Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb,
    float& error,
    float& count
    ) {
  const size_t BLK_SIZE = 32;
  size_t N = pc_m.w_*pc_m.h_;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,N/10,BLK_SIZE);
  ManagedDeviceImage<float> out(29,1);
  hipMemset(out.ptr_, 0, 29*sizeof(float));

  KernelICPStep<BLK_SIZE,D,Derived><<<blocks,threads,
    BLK_SIZE*sizeof(Vector29fda)>>>(
        pc_m,n_m,pc_o,n_o,T_mo,T_cm,cam,
        dotThr,distThr,10,out);
  checkCudaErrors(hipDeviceSynchronize());
  ManagedHostImage<float> sumAb(29,1);
  hipMemcpy(sumAb.ptr_,out.ptr_,29*sizeof(float), hipMemcpyDeviceToHost);

  //for (int i=0; i<29; ++i) std::cout << sumAb[i] << "\t";
  //std::cout << std::endl;
  ATA.fill(0.);
  ATb.fill(0.);
  int k = 0;
  for (int i=0; i<6; ++i) {
    for (int j=i; j<7; ++j) {
      float val = sumAb[k++];
      if (j==6)  {
        ATb(i) = val;
      } else {
        ATA(i,j) = val;
        ATA(j,i) = val;
      }
    }
  }
  count = sumAb[28];
  error = sumAb[27]/count;
  //std::cout << ATA << std::endl << ATb.transpose() << std::endl;
  //std::cout << "\terror&count " << error << " " << count << std::endl;
}

// explicit instantiation
template void ICPStep (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o, const SE3f& T_mo, const SE3f& T_cm,
    const CameraBase<float,Camera<float>::NumParams,Camera<float>>& cam,
    float dotThr, float distThr, Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb, float& error, float& count);
template void ICPStep (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o, const SE3f& T_mo, const SE3f& T_cm,
    const CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>>& cam,
    float dotThr, float distThr, Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb, float& error, float& count);

// T_mc: R_model_observation
template<int BLK_SIZE, int D, class Derived>
__global__ void KernelICPVisualizeAssoc(
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    SE3f T_mo,
    const CameraBase<float,D,Derived> cam,
    float dotThr,
    float distThr,
    int N_PER_T,
    Image<float> assoc_m,
    Image<float> assoc_o
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = idx*N_PER_T;
  const int N = pc_m.w_*pc_m.h_;
  const int idE = min(N,(idx+1)*N_PER_T);

  for (int id=idS; id<idE; ++id) {
    const int idx = id%pc_o.w_;
    const int idy = id/pc_o.w_;
    Vector3fda pc_oi = pc_o(idx,idy);
    // project current point into model frame to get association
    if (idx < pc_o.w_ && idy < pc_o.h_ && IsValidData(pc_oi)) {
      Vector3fda pc_o_in_m = T_mo * pc_oi;
      // project into model camera
      Vector2fda x_o_in_m = cam.Project(pc_o_in_m);
      int u = floor(x_o_in_m(0)+0.5f);
      int v = floor(x_o_in_m(1)+0.5f);
      if (0 <= u && u < pc_m.w_ && 0 <= v && v < pc_m.h_
          && pc_oi(2) > 0. && pc_o_in_m(2) > 0.
          && IsValidData(pc_o_in_m)) {
        // found association -> check thresholds;
        Vector3fda n_o_in_m = T_mo.rotation() * n_o(idx,idy);
        Vector3fda n_mi = n_m(u,v);
        Vector3fda pc_mi = pc_m(u,v);
        float dot  = n_mi.dot(n_o_in_m);
        float dist = (pc_mi-pc_o_in_m).norm();
        if (dot > dotThr && dist < distThr && IsValidData(pc_mi)) {
          // association is good -> accumulate
          //assoc_m(u,v) = n_mi.dot(-pc_mi+pc_o_in_m);
          //assoc_o(idx,idy) = n_mi.dot(-pc_mi+pc_o_in_m);
          assoc_m(u,v) = (-pc_mi+pc_o_in_m).norm();
          assoc_o(idx,idy) = (-pc_mi+pc_o_in_m).norm();
//          if (threadIdx.x < 3) printf("%d,%d and %d,%d\n", idx,idy,u,v);
        }
      }
    }
  }
}

template<int D, typename Derived>
void ICPVisualizeAssoc (
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    const SE3f& T_mo,
    const CameraBase<float,D,Derived>& cam,
    float angleThr,
    float distThr,
    Image<float>& assoc_m,
    Image<float>& assoc_o
    ) {
  size_t N = pc_m.w_*pc_m.h_;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,N/10,256);
  KernelICPVisualizeAssoc<256,D,Derived><<<blocks,threads>>>(pc_m,n_m,pc_o,n_o,
      T_mo,cam, cos(angleThr*M_PI/180.),distThr,10,assoc_m, assoc_o);
  checkCudaErrors(hipDeviceSynchronize());
}

template void ICPVisualizeAssoc (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o, const SE3f& T_mo,
    const CameraBase<float,Camera<float>::NumParams,Camera<float>>& cam,
    float angleThr, float distThr, Image<float>& assoc_m, Image<float>& assoc_o);
template void ICPVisualizeAssoc (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o, const SE3f& T_mo,
    const CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>>& cam,
    float angleThr, float distThr, Image<float>& assoc_m, Image<float>& assoc_o);

// T_mc: T_model_current
template<int BLK_SIZE>
__global__ void KernelICPStepRotation(
    Image<Vector3fda> n_m,
    Image<Vector3fda> n_o,
    Image<Vector3fda> pc_o,
    SE3f T_mo, 
    const Camera<float> cam,
    float dotThr,
    int N_PER_T,
    Image<float> out
    ) {
  assert(BLK_SIZE >=7);
  const int tid = threadIdx.x;
  const int id_ = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = id_*N_PER_T;
  const int idE = min((int)pc_o.Area(),(id_+1)*N_PER_T);
  SharedMemory<Vector7fda> smem;
  Vector7fda* sum = smem.getPointer();
  sum[tid] = Vector7fda::Zero();
  for (int id=idS; id<idE; ++id) {
    const int idx = id%pc_o.w_;
    const int idy = id/pc_o.w_;
    // project current point into model frame to get association
    if (idx >= pc_o.w_ || idy >= pc_o.h_) continue;
    Vector3fda pc_oi = pc_o(idx,idy);
    Vector3fda pc_o_in_m = T_mo * pc_oi ;
    // project into model camera
    // TODO: doing the association the other way around might be more
    // stable since the model depth is smoothed
    Vector2fda x_o_in_m = cam.Project(pc_o_in_m);
    const int u = floor(x_o_in_m(0)+0.5f);
    const int v = floor(x_o_in_m(1)+0.5f);
    if (0 <= u && u < pc_o.w_ && 0 <= v && v < pc_o.h_
        && pc_oi(2) > 0. && pc_o_in_m(2) > 0.
        && IsValidData(pc_o_in_m)) {
      // found association -> check thresholds;
      Vector3fda n_o_in_m = T_mo.rotation() * n_o(idx,idy);
      Vector3fda n_mi = n_m(u,v);
      const float dot  = n_mi.dot(n_o_in_m);
      if (dot > dotThr && IsValidData(n_mi)) {
        // association is good -> accumulate
        Eigen::Matrix<float,7,1,Eigen::DontAlign> upperTriangle;
        upperTriangle(0) = n_mi(0)*n_o_in_m(0);
        upperTriangle(1) = n_mi(1)*n_o_in_m(0);
        upperTriangle(2) = n_mi(2)*n_o_in_m(0);
        upperTriangle(3) = n_mi(1)*n_o_in_m(1);
        upperTriangle(4) = n_mi(1)*n_o_in_m(2);
        upperTriangle(5) = n_mi(2)*n_o_in_m(2);
        upperTriangle(6) = 1.; // to get number of data points
        sum[tid] += upperTriangle;
      }
    }
  }
  __syncthreads(); //sync the threads
#pragma unroll
  for(int s=(BLK_SIZE)/2; s>1; s>>=1) {
    if(tid < s) {
      sum[tid] += sum[tid+s];
    }
    __syncthreads();
  }
  if(tid < 7) {
    // sum the last two remaining matrixes directly into global memory
    atomicAdd(&out[tid], sum[0](tid)+sum[1](tid));
  }
}

void ICPStepRotation (
    Image<Vector3fda> n_m,
    Image<Vector3fda> n_o,
    Image<Vector3fda> pc_o,
    const SE3f& T_mo, 
    const Camera<float>& cam,
    float dotThr,
    Eigen::Matrix<float,3,3,Eigen::DontAlign>& N,
    float& count
    ) {
  const size_t BLK_SIZE = 32;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,pc_o.Area()/10,BLK_SIZE);
  ManagedDeviceImage<float> out(7,1);
  hipMemset(out.ptr_, 0, 7*sizeof(float));

  KernelICPStepRotation<BLK_SIZE><<<blocks,threads,
    BLK_SIZE*sizeof(Vector7fda)>>>(
        n_m,n_o,pc_o,T_mo,cam,
        dotThr,10,out);
  checkCudaErrors(hipDeviceSynchronize());
  ManagedHostImage<float> nUpperTri(7,1);
  hipMemcpy(nUpperTri.ptr_,out.ptr_,7*sizeof(float), hipMemcpyDeviceToHost);

  //for (int i=0; i<29; ++i) std::cout << sumAb[i] << "\t";
  //std::cout << std::endl;
  N.fill(0.);
  int k = 0;
  for (int i=0; i<3; ++i) {
    for (int j=i; j<3; ++j) {
      float val = nUpperTri[k++];
      N(i,j) = val;
      N(j,i) = val;
    }
  }
  count = nUpperTri[6];
  //std::cout << ATA << std::endl << ATb.transpose() << std::endl;
  //std::cout << "\terror&count " << error << " " << count << std::endl;
}

}
