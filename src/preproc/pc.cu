#include "hip/hip_runtime.h"

#include <tdp/cuda/hip/hip_runtime.h>
#include <tdp/nvidia/hip/hip_runtime_api.h>
#include <tdp/data/image.h>
#include <tdp/camera/camera_base.h>
#include <tdp/camera/camera.h>
#include <tdp/camera/camera_poly.h>
#include <tdp/eigen/dense.h>
#include <tdp/manifold/SE3.h>

namespace tdp {

template<int D, typename Derived>
__global__ void KernelDepth2PC(
    Image<float> d,
    CameraBase<float,D,Derived> cam,
    Image<Vector3fda> pc_c
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if (idx < pc_c.w_ && idy < pc_c.h_) {
    const float di = d(idx,idy);
    //if (100<idx&&idx<110 && 100<idy&&idy<110) printf("%f\n",di);
    if (di > 0) {
      pc_c(idx,idy) = cam.Unproject(idx,idy,di);
      //if (100<idx&&idx<110 && 100<idy&&idy<110) printf("%f %f %f\n",
      //    pc_c(idx,idy)(0),pc_c(idx,idy)(1),pc_c(idx,idy)(2));
    } else {
      pc_c(idx,idy)(0) = NAN; // nan
      pc_c(idx,idy)(1) = NAN; // nan
      pc_c(idx,idy)(2) = NAN; // nan
    }
  } else if (idx < d.w_ && idy < d.h_) {
    // d might be bigger than pc_c because of consecutive convolutions
    pc_c(idx,idy)(0) = NAN; // nan
    pc_c(idx,idy)(1) = NAN; // nan
    pc_c(idx,idy)(2) = NAN; // nan
  }
}

template<int D, typename Derived>
void Depth2PCGpu(
    const Image<float>& d,
    const CameraBase<float,D,Derived>& cam,
    Image<Vector3fda>& pc_c
    ) {

  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,d,32,32);
  KernelDepth2PC<D,Derived><<<blocks,threads>>>(d,cam,pc_c);
  checkCudaErrors(hipDeviceSynchronize());
}

template void Depth2PCGpu( const Image<float>& d,
    const CameraBase<float,Camera<float>::NumParams,Camera<float>>& cam,
    Image<Vector3fda>& pc_c);
template void Depth2PCGpu( const Image<float>& d,
    const CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>>& cam,
    Image<Vector3fda>& pc_c);

template<int D, typename Derived>
__global__ void KernelDepth2PC(
    Image<float> d,
    CameraBase<float,D,Derived> cam,
    SE3f T_rc,
    Image<Vector3fda> pc_r
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if (idx < pc_r.w_ && idy < pc_r.h_) {
    const float di = d(idx,idy);
    //if (100<idx&&idx<110 && 100<idy&&idy<110) printf("%f\n",di);
    if (di > 0) {
      //printf("%f",di);
      //printf("%f %f %f",cam.params_(0),cam.params_(1),cam.params_(2));
      pc_r(idx,idy) = T_rc*cam.Unproject(idx,idy,di);
      //if (100<idx&&idx<110 && 100<idy&&idy<110) printf("%f %f %f\n",
      //    pc_r(idx,idy)(0),pc_r(idx,idy)(1),pc_r(idx,idy)(2));
    } else {
      pc_r(idx,idy)(0) = NAN; // nan
      pc_r(idx,idy)(1) = NAN; // nan
      pc_r(idx,idy)(2) = NAN; // nan
    }
  } else if (idx < d.w_ && idy < d.h_) {
    // d might be bigger than pc_r because of consecutive convolutions
    pc_r(idx,idy)(0) = NAN; // nan
    pc_r(idx,idy)(1) = NAN; // nan
    pc_r(idx,idy)(2) = NAN; // nan
  }
}

template<int D, typename Derived>
void Depth2PCGpu(
    const Image<float>& d,
    const CameraBase<float,D,Derived>& cam,
    const SE3f& T_rc,
    Image<Vector3fda>& pc_r
    ) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,d,32,32);
  KernelDepth2PC<D,Derived><<<blocks,threads>>>(d,cam,T_rc,pc_r);
  checkCudaErrors(hipDeviceSynchronize());
}

// explicit instantiations
template void Depth2PCGpu(
    const Image<float>& d,
    const CameraBase<float,7,CameraPoly3<float>>& cam,
    const SE3f& T_rc,
    Image<Vector3fda>& pc_r
    );
template void Depth2PCGpu(
    const Image<float>& d,
    const CameraBase<float,Camera<float>::NumParams,Camera<float>>& cam,
    const SE3f& T_rc,
    Image<Vector3fda>& pc_r
    );


__global__ void KernelTransformPc(
    SE3f T_rc,
    Image<Vector3fda> pc_c
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if (idx < pc_c.w_ && idy < pc_c.h_) {
    pc_c(idx,idy) = T_rc*pc_c(idx,idy);
  }
}

void TransformPc(
    const SE3f& T_rc,
    Image<Vector3fda>& pc_c
    ) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,pc_c,32,32);
  KernelTransformPc<<<blocks,threads>>>(T_rc,pc_c);
  checkCudaErrors(hipDeviceSynchronize());
}

__global__ void KernelTransformPc(
    SO3f R_rc,
    Image<Vector3fda> pc_c
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if (idx < pc_c.w_ && idy < pc_c.h_) {
    pc_c(idx,idy) = R_rc*pc_c(idx,idy);
  }
}

void TransformPc(
    const SO3f& R_rc,
    Image<Vector3fda>& pc_c
    ) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,pc_c,32,32);
  KernelTransformPc<<<blocks,threads>>>(R_rc,pc_c);
  checkCudaErrors(hipDeviceSynchronize());
}

__global__ void KernelInverseTransformPc(
    SE3f T_rc,
    Image<Vector3fda> pc_c
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if (idx < pc_c.w_ && idy < pc_c.h_) {
    pc_c(idx,idy) = T_rc.Inverse()*pc_c(idx,idy);
  }
}

void InverseTransformPc(
    const SE3f& T_rc,
    Image<Vector3fda>& pc_c
    ) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,pc_c,32,32);
  KernelInverseTransformPc<<<blocks,threads>>>(T_rc,pc_c);
  checkCudaErrors(hipDeviceSynchronize());
}

//__global__ void KernelTransformPc(
//    SO3fda R_rc,
//    Image<Vector3fda> pc_c
//    ) {
//  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
//  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
//  if (idx < pc_c.w_ && idy < pc_c.h_) {
//    pc_c(idx,idy) = R_rc*pc_c(idx,idy);
//  }
//}
//
//void TransformPc(
//    const SO3fda& R_rc,
//    Image<Vector3fda>& pc_c
//    ) {
//  dim3 threads, blocks;
//  ComputeKernelParamsForImage(blocks,threads,pc_c,32,32);
//  KernelTransformPc<<<blocks,threads>>>(R_rc,pc_c);
//  checkCudaErrors(hipDeviceSynchronize());
//}
//
//__global__ void KernelInverseTransformPc(
//    SO3fda R_rc,
//    Image<Vector3fda> pc_c
//    ) {
//  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
//  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
////  Eigen::Matrix<float,3,3,Eigen::DontAlign> R = R_rc.Inverse().matrix();
//  if (idx < pc_c.w_ && idy < pc_c.h_) {
////    if (idx == 0 && idy == 0) {
////      Eigen::Quaternion<float,Eigen::DontAlign> q(R_rc.vector());
////      printf("q: %f %f %f %f", q.w(), q.x(), q.y(), q.z());
////      q = Eigen::Quaternion<float,Eigen::DontAlign>(R_rc.Inverse().vector());
////      printf("q: %f %f %f %f", q.w(), q.x(), q.y(), q.z());
////    }
////    pc_c(idx,idy) = R*pc_c(idx,idy);
//    pc_c(idx,idy) = R_rc.Inverse()*pc_c(idx,idy);
////    Vector3fda p = R_rc.Inverse()*pc_c(idx,idy);
////    pc_c(idx,idy) = p;
//  }
//}
//
//void InverseTransformPc(
//    const SO3fda& R_rc,
//    Image<Vector3fda>& pc_c
//    ) {
//  dim3 threads, blocks;
//  ComputeKernelParamsForImage(blocks,threads,pc_c,32,32);
//  KernelInverseTransformPc<<<blocks,threads>>>(R_rc,pc_c);
//  checkCudaErrors(hipDeviceSynchronize());
//}

__global__ 
void KernelL2Distance(Image<Vector3fda> pcA, Image<Vector3fda> pcB,
    SE3f T_ab,
    Image<float> dist) {
  //const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if (idx < pcA.w_ && idy < pcA.h_) {
    dist(idx,idy) = (pcA(idx,idy)-T_ab*pcB(idx,idy)).norm();
  }
}

void L2Distance(
    const Image<Vector3fda>& pcA,
    const Image<Vector3fda>& pcB,
    const SE3f& T_ab,
    Image<float>& dist
    ) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,pcA,32,32);
  KernelL2Distance<<<blocks,threads>>>(pcA,pcB,T_ab,dist);
  checkCudaErrors(hipDeviceSynchronize());
}

}
