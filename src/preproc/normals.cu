#include "hip/hip_runtime.h"

#include <tdp/eigen/dense.h>
#include <tdp/cuda/hip/hip_runtime.h>
#include <tdp/data/image.h>
#include <tdp/preproc/normals.h>
#include <tdp/nvidia/hip/hip_runtime_api.h>
#include <tdp/manifold/SO3.h>

namespace tdp {

__global__ 
void KernelSurfaceNormals(Image<float> d,
    Image<float> ddu, Image<float> ddv,
    Image<Vector3fda> n, float f, float uc, float vc) {
  //const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if (idx < n.w_ && idy < n.h_) {
    const float di = d(idx,idy);
    float* ni = (float*)(&(n(idx,idy)));
    if (di > 0) {
      const float ddui = ddu(idx,idy);
      const float ddvi = ddv(idx,idy);
      ni[0] = -ddui*f;
      ni[1] = -ddvi*f;
      ni[2] = ((idx-uc)*ddui + (idy-vc)*ddvi + di);
      const float norm = sqrtf(ni[0]*ni[0] + ni[1]*ni[1] + ni[2]*ni[2]);
      ni[0] /= norm;
      ni[1] /= norm;
      ni[2] /= norm;
    } else {
      ni[0] = NAN;
      ni[1] = NAN;
      ni[2] = NAN;
    }
  }
}

void ComputeNormals(
    const Image<float>& d,
    const Image<float>& ddu,
    const Image<float>& ddv,
    const Image<Vector3fda>& n,
    float f, float uc, float vc) {
  
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,d,32,32);
  KernelSurfaceNormals<<<blocks,threads>>>(d,ddu,ddv,n,f,uc,vc);
  checkCudaErrors(hipDeviceSynchronize());
}

__global__ 
void KernelSurfaceNormals(Image<float> d,
    Image<float> ddu, Image<float> ddv,
    Image<Vector3fda> n, 
    SE3f T_rc,
    float f, float uc, float vc) {
  //const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if (idx < n.w_ && idy < n.h_) {
    const float di = d(idx,idy);
    float* ni = (float*)(&(n(idx,idy)));
    if (di > 0) {
      const float ddui = ddu(idx,idy);
      const float ddvi = ddv(idx,idy);
      ni[0] = -ddui*f;
      ni[1] = -ddvi*f;
      ni[2] = ((idx-uc)*ddui + (idy-vc)*ddvi + di);
      const float norm = sqrtf(ni[0]*ni[0] + ni[1]*ni[1] + ni[2]*ni[2]);
      ni[0] /= norm;
      ni[1] /= norm;
      ni[2] /= norm;
      n(idx,idy) = T_rc.rotation()*n(idx,idy);
    } else {
      ni[0] = NAN;
      ni[1] = NAN;
      ni[2] = NAN;
    }
  }
}

void ComputeNormals(
    const Image<float>& d,
    const Image<float>& ddu,
    const Image<float>& ddv,
    const Image<Vector3fda>& n,
    const SE3f& T_rc,
    float f, float uc, float vc) {
  
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,d,32,32);
  KernelSurfaceNormals<<<blocks,threads>>>(d,ddu,ddv,n,T_rc,f,uc,vc);
  checkCudaErrors(hipDeviceSynchronize());
}

__global__ 
void KernelSurfaceNormals2Image(
    Image<Vector3fda> n, Image<Vector3bda> n2d) {
  //const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if (idx < n.w_ && idy < n.h_) {
    Vector3fda ni = n(idx,idy);
    if (IsValidNormal(ni)) {
      n2d(idx,idy)(0) = floor(ni(0)*128+127);
      n2d(idx,idy)(1) = floor(ni(1)*128+127);
      n2d(idx,idy)(2) = floor(ni(2)*128+127);
    } else {
      n2d(idx,idy)(0) = 0;
      n2d(idx,idy)(1) = 0;
      n2d(idx,idy)(2) = 0;
    }
  }
}

void Normals2Image(
    const Image<Vector3fda>& n,
    Image<Vector3bda>& n2d
    ) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,n,32,32);
  KernelSurfaceNormals2Image<<<blocks,threads>>>(n,n2d);
  checkCudaErrors(hipDeviceSynchronize());
}

__global__ 
void KernelRenormalizeSurfaceNormals(Image<Vector3fda> n, float normThr) {
  //const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if (idx < n.w_ && idy < n.h_) {
    Vector3fda ni = n(idx,idy);
    if (IsValidData(ni)) {
      float norm = ni.norm();
      if (norm < normThr) {
        norm = 0.f;
      }
      n(idx,idy) = ni / norm;
    }
  }
}

void RenormalizeSurfaceNormals(
    Image<Vector3fda>& n,
    float normThr
    ) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,n,32,32);
  KernelRenormalizeSurfaceNormals<<<blocks,threads>>>(n, normThr);
  checkCudaErrors(hipDeviceSynchronize());
}

__global__ 
void KernelAngularDeviation(Image<Vector3fda> nA, Image<Vector3fda> nB,
    SO3f T_ab,
    Image<float> ang) {
  //const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if (idx < nA.w_ && idy < nA.h_) {
    ang(idx,idy) = acos(min(1.f,max(-1.f,nA(idx,idy).dot(T_ab*nB(idx,idy)))));
  }
}

void AngularDeviation(
    const Image<Vector3fda>& nA,
    const Image<Vector3fda>& nB,
    const SO3f& T_ab,
    Image<float>& ang
    ) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,nA,32,32);
  KernelAngularDeviation<<<blocks,threads>>>(nA,nB,T_ab,ang);
  checkCudaErrors(hipDeviceSynchronize());
}

}
