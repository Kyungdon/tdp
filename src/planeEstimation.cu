#include "hip/hip_runtime.h"

#include <Eigen/Dense>
#include <tdp/image.h>
#include <tdp/calibration/planeEstimation.h>
#include <tdp/camera.h>
#include <tdp/nvidia/hip/hip_runtime_api.h>

namespace tdp {

__host__ __device__
float HuberCost(float x, float alpha) {
  float absx = fabs(x);
  return absx <= alpha ? 0.5*x*x : alpha*(absx - 0.5*alpha);
}

__host__ __device__
float HuberCostDeriv(float x, float alpha) {
  return (x < alpha ? - alpha : (x > alpha ? alpha : x));
}

__global__
void KernelPlaneEstimationHuberDeriv(
    Image<float> d,
    Camera<float> cam,
    Eigen::Vector3f nd,
    float alpha,
    Image<float> f,
    Image<Eigen::Vector3f> deriv
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if (idx < d.w_ && idy < d.h_) {
    float di = d(idx,idy);
    if (di==di) {
      Eigen::Vector3f p = cam.Unproject(idx,idy,di);
      float xi = nd.dot(p)+1;
      f(idx,idy) = HuberCost(xi,alpha);
      deriv(idx,idy) = HuberCostDeriv(xi,alpha)*p;
    } else {
      f(idx,idy) = 0./0.;
      deriv(idx,idy)(0) = 0./0.;
      deriv(idx,idy)(1) = 0./0.;
      deriv(idx,idy)(2) = 0./0.;
    }
  }
}

void PlaneEstimationHuberDeriv(
    const Image<float>& d,
    const Camera<float>& cam,
    const Eigen::Vector3f& nd,
    float alpha,
    Image<float>& f,
    Image<Eigen::Vector3f>& deriv) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,d,32,32);
  KernelPlaneEstimationHuberDeriv<<<blocks,threads>>>(d,cam,nd,alpha,f,deriv);
  checkCudaErrors(hipDeviceSynchronize());
}

}
