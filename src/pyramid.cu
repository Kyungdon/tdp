#include "hip/hip_runtime.h"
#include <assert.h>
#include 

template<typename T>
__global__
void KernelPyrDown(
    const Image<T>& Iin,
    Image<T>& Iout,
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if (idx < Iout.w_ && idy < Iout.h_) {
    T* in0 = Iin.RowPtr(idy*2);
    T* in1 = Iin.RowPtr(idy*2+1);
    Iout(idx,idy) = 0.25f*(in0[idx*2] + in0[idx*2+1] + in1[idx*2] + in1[idx*2+1]);
  }
}

void PyrDown(
    const Image<float>& Iin,
    Image<float>& Iout,
    ) {
  assert(Iin.w_ == Iout.w_*2);
  assert(Iin.h_ == Iout.h_*2);
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,Iout,32,32);
  KernelPyrDown<float><<<blocks,threads>>>(Iin,Iout);
  checkCudaErrors(hipDeviceSynchronize());
}
