#include "hip/hip_runtime.h"
#include <tdp/nn_cuda/nn_cuda.h>
#include <tdp/sorts/parallelSorts.h>
#include <tdp/cuda/hip/hip_runtime.h>

namespace tdp {

template<class T> inline void destroyArray(T*& p) {
  if (p) {
    delete[] p;
    p = nullptr;
  }
}

inline void NN_Cuda::clearHostMemory() {
  destroyArray(h_points);
  destroyArray(h_elements);
}

template<class T> inline void destroyDevicePointer(T*& p) {
  if (p) {
    hipFree(p);
    p = nullptr;
  }
}

inline void NN_Cuda::clearDeviceMemory() {
  destroyDevicePointer(d_points);
  destroyDevicePointer(d_elements);
}

inline void NN_Cuda::clearMemory() {
  clearHostMemory();
  clearDeviceMemory();
}

NN_Cuda::~NN_Cuda() {
  clearMemory();
}

void NN_Cuda::reinitialise(Image<Vector3fda>& pc, int stride) {
  // Reset this object
  clearMemory();

  initHostMemory(pc, stride);
  initDeviceMemory();
}

void NN_Cuda::initHostMemory(Image<Vector3fda>&pc, int stride) {
  // Copy all of the points into this nearest neighbor buffer
  m_size = pc.Area();
  h_points = new Vector3fda[m_size];
  for (size_t index = 0; index < pc.Area(); index += stride) {
    h_points[index] = pc[index];
  }
  h_elements = new NN_Element[m_size];
}

void NN_Cuda::initDeviceMemory() {
  // Initialize the device memory as necessary
  hipMalloc(&d_points, m_size * sizeof(Vector3fda));
  hipMalloc(&d_elements, m_size * sizeof(NN_Element));

  // Copy the points into the device
  hipMemcpy(d_points, h_points, m_size * sizeof(Vector3fda), hipMemcpyHostToDevice);
}

__global__
void KernelComputeNNDistances(
     size_t numElements,
     NN_Element* elements,
     Vector3fda* points,
     Vector3fda query
) {
  size_t index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < numElements) {
    Vector3fda diff = points[index] - query;
    elements[index] = NN_Element(diff.dot(diff), index);
  }
}


void NN_Cuda::search(
     Vector3fda& query,
     int k,
     Eigen::VectorXi& nnIds,
     Eigen::VectorXf& dists
) const {
  // compute the distances for every point
  dim3 blocks, threads;
  ComputeKernelParamsForArray(blocks, threads, m_size, 256);
  KernelComputeNNDistances<<<blocks,threads>>>(m_size, d_elements, d_points, query);

  // Sort nearest to farthest
  ParallelSorts<NN_Element>::sortDevicePreloaded(blocks, threads, m_size, d_elements);

  // Copy Back data
  hipMemcpy(h_elements, d_elements, m_size * sizeof(NN_Element), hipMemcpyDeviceToHost);

  // Place the necessary information into the passed containers
  for (size_t i = 0; i < k; i++) {
    nnIds(i) = h_elements[i].index();
    dists(i) = h_elements[i].value();
  }
}

}
