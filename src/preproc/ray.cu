#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <Eigen/Core>
#include <Eigen/Dense>
#include <tdp/data/image.h>
#include <tdp/eigen/dense.h>
#include <tdp/camera/camera_base.h>

namespace tdp {

// populate rays given a pinhole camera parameterization
template<int D, typename Derived>
__global__ 
void KernelRay(
    CameraBase<float,D,Derived> cam,
    Image<Vector3fda> ray) {

  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if (idx < ray.w_ && idy < ray.h_) {
    ray(idx, idy) = cam.Unproject(idx,idy,1.);
  }
}

template<int D, typename Derived>
void ComputeCameraRays(
    const CameraBase<float,D,Derived>& cam,
    Image<Vector3fda>& ray 
    ) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,ray,32,32);
  KernelDepth2PC<D,Derived><<<blocks,threads>>>(cam,ray);
  checkCudaErrors(hipDeviceSynchronize());
}

template void ComputeCameraRays(
    const CameraBase<float,Camera<float>::NumParams,Camera<float>>& cam,
    Image<Vector3fda>& ray 
    );
template void ComputeCameraRays(
    const CameraBaseCameraf& cam,
    Image<Vector3fda>& ray 
    );
template void ComputeCameraRays(
    const CameraBaseCameraPoly3f& cam,
    Image<Vector3fda>& ray 
    );

}
