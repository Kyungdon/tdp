#include "hip/hip_runtime.h"

#include <Eigen/Dense>
#include <tdp/data/image.h>
#include <tdp/eigen/dense.h>
#include <tdp/calibration/planeEstimation.h>
#include <tdp/camera/camera.h>
#include <tdp/nvidia/hip/hip_runtime_api.h>

namespace tdp {

__host__ __device__
float HuberCost(float x, float alpha) {
  float absx = fabs(x);
  return absx <= alpha ? 0.5*x*x : alpha*(absx - 0.5*alpha);
}

__host__ __device__
float HuberCostDeriv(float x, float alpha) {
  return (x < alpha ? - alpha : (x > alpha ? alpha : x));
}

__global__
void KernelPlaneEstimationHuberDeriv(
    Image<float> d,
    Camera<float> cam,
    Vector3fda nd,
    float alpha,
    Image<float> f,
    Image<Vector3fda> deriv
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if (idx < d.w_ && idy < d.h_) {
    float di = d(idx,idy);
    if (di==di) {
      Vector3fda p = cam.Unproject(idx,idy,di);
      float xi = nd.dot(p)+1;
      f(idx,idy) = HuberCost(xi,alpha);
      deriv(idx,idy) = HuberCostDeriv(xi,alpha)*p;
    } else {
      f(idx,idy) = 0./0.;
      deriv(idx,idy)(0) = 0./0.;
      deriv(idx,idy)(1) = 0./0.;
      deriv(idx,idy)(2) = 0./0.;
    }
  }
}

void PlaneEstimationHuberDeriv(
    const Image<float>& d,
    const Camera<float>& cam,
    const Vector3fda& nd,
    float alpha,
    Image<float>& f,
    Image<Vector3fda>& deriv) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,d,32,32);
  KernelPlaneEstimationHuberDeriv<<<blocks,threads>>>(d,cam,nd,alpha,f,deriv);
  checkCudaErrors(hipDeviceSynchronize());
}

}
