#include "hip/hip_runtime.h"
/* Copyright (c) 2016, Julian Straub <jstraub@csail.mit.edu> Licensed
 * under the MIT license. See the license file LICENSE.
 */

#include <assert.h>
#include <tdp/eigen/dense.h>
#include <tdp/cuda/hip/hip_runtime.h>
#include <tdp/nvidia/hip/hip_runtime_api.h>
#include <tdp/data/image.h>
#include <tdp/data/managed_image.h>
#include <tdp/camera/camera.h>
#include <tdp/camera/camera_poly.h>
#include <tdp/reductions/reductions.cuh>
#include <tdp/manifold/SE3.h>
#include <tdp/cuda/cuda.cuh>
#include <tdp/icp/icp.cuh>

namespace tdp {

// T_mc: R_model_observation
template<int BLK_SIZE>
__global__ void KernelICPStep(
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    Image<int> assoc_om,
    SE3f T_mo, 
    float dotThr,
    float distThr,
    int N_PER_T,
    Image<float> out
    ) {
  assert(BLK_SIZE >=29);
  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = idx*N_PER_T;
  const int N = pc_m.w_*pc_m.h_;
  const int idE = min(N,(idx+1)*N_PER_T);

  SharedMemory<Vector29fda> smem;
  Vector29fda* sum = smem.getPointer();

  sum[tid] = Vector29fda::Zero();
  for (int id=idS; id<idE; ++id) {
    const int x = id%pc_m.w_;
    const int y = id/pc_m.w_;
    if (x<pc_m.w_ && y<pc_m.h_) {
      const int id_o = assoc_om(x,y);
      const int u = id_o%pc_o.w_;
      const int v = id_o/pc_o.w_;
      if (0<=u && u<pc_o.w_ && 0<=v && v<pc_o.h_) {
        // found association -> check thresholds;
        Vector3fda n_o_in_m = T_mo.rotation()*n_o(u,v);
        Vector3fda n_mi = n_m(x,y);
        Vector3fda pc_mi = pc_m(x,y);
        Vector3fda pc_oi = pc_o(u,v);
        Vector3fda pc_o_in_m = T_mo * pc_oi;
        const float dot  = n_mi.dot(n_o_in_m);
        const float dist = (pc_mi-pc_o_in_m).norm();
        if (dot > dotThr && dist < distThr && IsValidData(pc_mi)) {
          // association is good -> accumulate
          // if we found a valid association accumulate the A and b for A x = b
          // where x \in se{3} as well as the residual error
          float ab[7];      
          Eigen::Map<Vector3fda> top(&(ab[0]));
          Eigen::Map<Vector3fda> bottom(&(ab[3]));
          // as in mp3guy: 
          top = (pc_o_in_m).cross(n_mi);
          bottom = n_mi;
          ab[6] = n_mi.dot(pc_mi-pc_o_in_m);
          Eigen::Matrix<float,29,1,Eigen::DontAlign> upperTriangle;
          int k=0;
#pragma unroll
          for (int i=0; i<7; ++i) {
            for (int j=i; j<7; ++j) {
              upperTriangle(k++) = ab[i]*ab[j];
            }
          }
          upperTriangle(28) = 1.; // to get number of data points
          sum[tid] += upperTriangle;
        }
      }
    }
  }
  __syncthreads(); //sync the threads
#pragma unroll
  for(int s=(BLK_SIZE)/2; s>1; s>>=1) {
    if(tid < s) {
      sum[tid] += sum[tid+s];
    }
    __syncthreads();
  }
  if(tid < 29) {
    // sum the last two remaining matrixes directly into global memory
    atomicAdd(&out[tid], sum[0](tid)+sum[1](tid));
  }
}

void ICPStep (
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    Image<int> assoc_om,
    const SE3f& T_mo, 
    float dotThr,
    float distThr,
    Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb,
    float& error,
    float& count
    ) {
  const size_t BLK_SIZE = 32;
  size_t N = pc_m.w_*pc_m.h_;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,N/10,BLK_SIZE);
  ManagedDeviceImage<float> out(29,1);
  hipMemset(out.ptr_, 0, 29*sizeof(float));

  KernelICPStep<BLK_SIZE><<<blocks,threads,
    BLK_SIZE*sizeof(Vector29fda)>>>(
        pc_m,n_m,pc_o,n_o,assoc_om,T_mo,dotThr,distThr,10,out);
  checkCudaErrors(hipDeviceSynchronize());
  ManagedHostImage<float> sumAb(29,1);
  hipMemcpy(sumAb.ptr_,out.ptr_,29*sizeof(float), hipMemcpyDeviceToHost);

  //for (int i=0; i<29; ++i) std::cout << sumAb[i] << "\t";
  //std::cout << std::endl;
  ATA.fill(0.);
  ATb.fill(0.);
  int k = 0;
  for (int i=0; i<6; ++i) {
    for (int j=i; j<7; ++j) {
      float val = sumAb[k++];
      if (j==6)  {
        ATb(i) = val;
      } else {
        ATA(i,j) = val;
        ATA(j,i) = val;
      }
    }
  }
  count = sumAb[28];
  error = sumAb[27]/count;
  //std::cout << ATA << std::endl << ATb.transpose() << std::endl;
  //std::cout << "\terror&count " << error << " " << count << std::endl;
}

// T_mc: R_model_observation
template<int BLK_SIZE, int D, typename Derived>
__global__ void KernelICPStep(
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    SE3f T_mo, 
    SE3f T_co, 
    const CameraBase<float,D,Derived> cam,
    float dotThr,
    float distThr,
    int N_PER_T,
    Image<float> out
    ) {
  assert(BLK_SIZE >=29);
  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = idx*N_PER_T;
  const int N = pc_m.w_*pc_m.h_;
  const int idE = min(N,(idx+1)*N_PER_T);

  SharedMemory<Vector29fda> smem;
  Vector29fda* sum = smem.getPointer();

  sum[tid] = Vector29fda::Zero();
  for (int id=idS; id<idE; ++id) {
    const int x = id%pc_o.w_;
    const int y = id/pc_o.w_;
    int u, v;
    //printf("%d %d\n",x,y);
    int res = AssociateModelIntoCurrent<D,Derived>(x, y, pc_m, T_mo,
        T_co, cam, u, v);
    //printf("%d %d %d\n",x,y,res);
    if (res == 0) {
      //printf("%d %d %d %d\n",x,y,u,v);
      // found association -> check thresholds;
      Vector3fda n_o_in_m = T_mo.rotation()*n_o(u,v);
      Vector3fda n_mi = n_m(x,y);
      Vector3fda n_m_in_oi = T_mo.rotation().Inverse()*n_mi;
      Vector3fda pc_mi = pc_m(x,y);
      Vector3fda pc_oi = pc_o(u,v);
      Vector3fda pc_o_in_m = T_mo * pc_oi;
      const float dot  = n_mi.dot(n_o_in_m);
      const float dist = (pc_mi-pc_o_in_m).norm();
      if (dot > dotThr && dist < distThr && IsValidData(pc_mi)) {
        // association is good -> accumulate
        // if we found a valid association accumulate the A and b for A x = b
        // where x \in se{3} as well as the residual error
        float ab[7];      
        Eigen::Map<Vector3fda> top(&(ab[0]));
        Eigen::Map<Vector3fda> bottom(&(ab[3]));
        // as in mp3guy:  (left multiplication of error)
//        top = (pc_o_in_m).cross(n_mi);
//        bottom = n_mi;
        // right multiplication of error
        top = (pc_oi).cross(n_m_in_oi);
        bottom = n_m_in_oi;
        float nTt = n_mi.dot(T_mo.translation());
//        top.array() += nTt;
//        bottom.array() += nTt;
        ab[6] = n_mi.dot(pc_mi-pc_o_in_m);
        Eigen::Matrix<float,29,1,Eigen::DontAlign> upperTriangle;
        int k=0;
#pragma unroll
        for (int i=0; i<7; ++i) {
          for (int j=i; j<7; ++j) {
            upperTriangle(k++) = ab[i]*ab[j];
          }
        }
        upperTriangle(28) = 1.; // to get number of data points
        sum[tid] += upperTriangle;
      }
    }
  }
  __syncthreads(); //sync the threads
#pragma unroll
  for(int s=(BLK_SIZE)/2; s>1; s>>=1) {
    if(tid < s) {
      sum[tid] += sum[tid+s];
    }
    __syncthreads();
  }
  if(tid < 29) {
    // sum the last two remaining matrixes directly into global memory
    atomicAdd(&out[tid], sum[0](tid)+sum[1](tid));
  }
}

template<int D, typename Derived>
void ICPStep (
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    const SE3f& T_mo, 
    const SE3f& T_cm,
    const CameraBase<float,D,Derived>& cam,
    float dotThr,
    float distThr,
    Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb,
    float& error,
    float& count
    ) {
  const size_t BLK_SIZE = 32;
  size_t N = pc_m.w_*pc_m.h_;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,N/10,BLK_SIZE);
  ManagedDeviceImage<float> out(29,1);
  hipMemset(out.ptr_, 0, 29*sizeof(float));

  KernelICPStep<BLK_SIZE,D,Derived><<<blocks,threads,
    BLK_SIZE*sizeof(Vector29fda)>>>(
        pc_m,n_m,pc_o,n_o,T_mo,T_cm,cam,
        dotThr,distThr,10,out);
  checkCudaErrors(hipDeviceSynchronize());
  ManagedHostImage<float> sumAb(29,1);
  hipMemcpy(sumAb.ptr_,out.ptr_,29*sizeof(float), hipMemcpyDeviceToHost);

  //for (int i=0; i<29; ++i) std::cout << sumAb[i] << "\t";
  //std::cout << std::endl;
  ATA.fill(0.);
  ATb.fill(0.);
  int k = 0;
  for (int i=0; i<6; ++i) {
    for (int j=i; j<7; ++j) {
      float val = sumAb[k++];
      if (j==6)  {
        ATb(i) = val;
      } else {
        ATA(i,j) = val;
        ATA(j,i) = val;
      }
    }
  }
  count = sumAb[28];
  error = sumAb[27]/count;
  //std::cout << ATA << std::endl << ATb.transpose() << std::endl;
  //std::cout << "\terror&count " << error << " " << count << std::endl;
}

// explicit instantiation
template void ICPStep (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o, const SE3f& T_mo, const SE3f& T_cm,
    const CameraBase<float,Camera<float>::NumParams,Camera<float>>& cam,
    float dotThr, float distThr, Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb, float& error, float& count);
template void ICPStep (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o, const SE3f& T_mo, const SE3f& T_cm,
    const CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>>& cam,
    float dotThr, float distThr, Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb, float& error, float& count);


// T_mc: R_model_observation
template<int BLK_SIZE, int D, class Derived>
__global__ void KernelICPVisualizeAssoc(
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    SE3f T_mo,
    const CameraBase<float,D,Derived> cam,
    float dotThr,
    float distThr,
    int N_PER_T,
    Image<float> assoc_m,
    Image<float> assoc_o
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = idx*N_PER_T;
  const int N = pc_m.w_*pc_m.h_;
  const int idE = min(N,(idx+1)*N_PER_T);

  for (int id=idS; id<idE; ++id) {
    const int x = id%pc_m.w_;
    const int y = id/pc_m.w_;
    int u, v;
    int res = AssociateModelIntoCurrent<D,Derived>(x, y, pc_m, T_mo,
        tdp::SE3f(), cam, u, v);
    if (res == 0) {
      // found association -> check thresholds;
      Vector3fda pc_mi = pc_m(x,y);
      Vector3fda n_mi = n_m(x,y);
      Vector3fda n_o_in_m = T_mo.rotation() * n_o(u,v);
      Vector3fda pc_oi = pc_o(u,v);
      Vector3fda pc_o_in_m = T_mo * pc_oi;
      float dot  = n_mi.dot(n_o_in_m);
      float dist = (pc_mi-pc_o_in_m).norm();
      if (dot > dotThr && dist < distThr && IsValidData(pc_mi)) {
        // association is good -> accumulate
        //assoc_m(u,v) = n_mi.dot(-pc_mi+pc_o_in_m);
        //assoc_o(x,y) = n_mi.dot(-pc_mi+pc_o_in_m);
        assoc_m(x,y) = n_mi.dot(-pc_mi+pc_o_in_m);
        //        assoc_o(u,v) = n_mi.dot(-pc_mi+pc_o_in_m);
        //          if (threadIdx.x < 3) printf("%d,%d and %d,%d\n", x,y,u,v);
      }
    } else if (res < 3) {
      assoc_o(x,y) = res;
    }
  }
}

template<int D, typename Derived>
void ICPVisualizeAssoc (
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    const SE3f& T_mo,
    const CameraBase<float,D,Derived>& cam,
    float angleThr,
    float distThr,
    Image<float>& assoc_m,
    Image<float>& assoc_o
    ) {
  size_t N = pc_m.w_*pc_m.h_;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,N/10,256);
  KernelICPVisualizeAssoc<256,D,Derived><<<blocks,threads>>>(pc_m,n_m,pc_o,n_o,
      T_mo,cam, cos(angleThr*M_PI/180.),distThr,10,assoc_m, assoc_o);
  checkCudaErrors(hipDeviceSynchronize());
}

template void ICPVisualizeAssoc (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o, const SE3f& T_mo,
    const CameraBase<float,Camera<float>::NumParams,Camera<float>>& cam,
    float angleThr, float distThr, Image<float>& assoc_m, Image<float>& assoc_o);
template void ICPVisualizeAssoc (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o, const SE3f& T_mo,
    const CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>>& cam,
    float angleThr, float distThr, Image<float>& assoc_m, Image<float>& assoc_o);


}
