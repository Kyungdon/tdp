#include "hip/hip_runtime.h"
/* Copyright (c) 2016, Julian Straub <jstraub@csail.mit.edu> Licensed
 * under the MIT license. See the license file LICENSE.
 */

#include <assert.h>
#include <tdp/eigen/dense.h>
#include <tdp/cuda/hip/hip_runtime.h>
#include <tdp/nvidia/hip/hip_runtime_api.h>
#include <tdp/data/image.h>
#include <tdp/data/managed_image.h>
#include <tdp/camera/camera.h>
#include <tdp/camera/camera_poly.h>
#include <tdp/reductions/reductions.cuh>
#include <tdp/manifold/SE3.h>
#include <tdp/cuda/cuda.cuh>

namespace tdp {

template<int D, class Derived>
__device__ 
inline int AssociateModelIntoCurrent(
    int x, int y, 
    const Image<Vector3fda>& pc_m,
    const SE3f& T_mo,
    const SE3f& T_co,
    const CameraBase<float,D,Derived>& cam,
    int& u, int& v
    ) {
  // project model point into camera frame to get association
  if (x < pc_m.w_ && y < pc_m.h_ ) {
    Vector3fda pc_mi = pc_m(x,y);
    if (IsValidData(pc_mi)) {
      Vector3fda pc_m_in_o = T_mo.Inverse() * pc_mi;
      // project into current camera
      Vector2fda x_m_in_o = cam.Project(T_co*pc_m_in_o);
      u = floor(x_m_in_o(0)+0.5f);
      v = floor(x_m_in_o(1)+0.5f);
      if (0 <= u && u < pc_m.w_ && 0 <= v && v < pc_m.h_
          && pc_m_in_o(2) > 0.
          && IsValidData(pc_m_in_o)) {
        return 0;
      } else {
        return 1;
      }
    } else {
      return 2;
    }
  } else {
    return 3;
  }
}


// T_mc: R_model_observation
template<int BLK_SIZE>
__global__ void KernelICPStep(
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    Image<int> assoc_om,
    SE3f T_mo, 
    float dotThr,
    float distThr,
    int N_PER_T,
    Image<float> out
    ) {
  assert(BLK_SIZE >=29);
  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = idx*N_PER_T;
  const int N = pc_m.w_*pc_m.h_;
  const int idE = min(N,(idx+1)*N_PER_T);

  SharedMemory<Vector29fda> smem;
  Vector29fda* sum = smem.getPointer();

  sum[tid] = Vector29fda::Zero();
  for (int id=idS; id<idE; ++id) {
    const int x = id%pc_m.w_;
    const int y = id/pc_m.w_;
    if (x<pc_m.w_ && y<pc_m.h_) {
      const int id_o = assoc_om(x,y);
      const int u = id_o%pc_o.w_;
      const int v = id_o/pc_o.w_;
      if (0<=u && u<pc_o.w_ && 0<=v && v<pc_o.h_) {
        // found association -> check thresholds;
        Vector3fda n_o_in_m = T_mo.rotation()*n_o(u,v);
        Vector3fda n_mi = n_m(x,y);
        Vector3fda pc_mi = pc_m(x,y);
        Vector3fda pc_oi = pc_o(u,v);
        Vector3fda pc_o_in_m = T_mo * pc_oi;
        const float dot  = n_mi.dot(n_o_in_m);
        const float dist = (pc_mi-pc_o_in_m).norm();
        if (dot > dotThr && dist < distThr && IsValidData(pc_mi)) {
          // association is good -> accumulate
          // if we found a valid association accumulate the A and b for A x = b
          // where x \in se{3} as well as the residual error
          float ab[7];      
          Eigen::Map<Vector3fda> top(&(ab[0]));
          Eigen::Map<Vector3fda> bottom(&(ab[3]));
          // as in mp3guy: 
          top = (pc_o_in_m).cross(n_mi);
          bottom = n_mi;
          ab[6] = n_mi.dot(pc_mi-pc_o_in_m);
          Eigen::Matrix<float,29,1,Eigen::DontAlign> upperTriangle;
          int k=0;
#pragma unroll
          for (int i=0; i<7; ++i) {
            for (int j=i; j<7; ++j) {
              upperTriangle(k++) = ab[i]*ab[j];
            }
          }
          upperTriangle(28) = 1.; // to get number of data points
          sum[tid] += upperTriangle;
        }
      }
    }
  }
  __syncthreads(); //sync the threads
#pragma unroll
  for(int s=(BLK_SIZE)/2; s>1; s>>=1) {
    if(tid < s) {
      sum[tid] += sum[tid+s];
    }
    __syncthreads();
  }
  if(tid < 29) {
    // sum the last two remaining matrixes directly into global memory
    atomicAdd(&out[tid], sum[0](tid)+sum[1](tid));
  }
}

void ICPStep (
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    Image<int> assoc_om,
    const SE3f& T_mo, 
    float dotThr,
    float distThr,
    Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb,
    float& error,
    float& count
    ) {
  const size_t BLK_SIZE = 32;
  size_t N = pc_m.w_*pc_m.h_;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,N/10,BLK_SIZE);
  ManagedDeviceImage<float> out(29,1);
  hipMemset(out.ptr_, 0, 29*sizeof(float));

  KernelICPStep<BLK_SIZE><<<blocks,threads,
    BLK_SIZE*sizeof(Vector29fda)>>>(
        pc_m,n_m,pc_o,n_o,assoc_om,T_mo,dotThr,distThr,10,out);
  checkCudaErrors(hipDeviceSynchronize());
  ManagedHostImage<float> sumAb(29,1);
  hipMemcpy(sumAb.ptr_,out.ptr_,29*sizeof(float), hipMemcpyDeviceToHost);

  //for (int i=0; i<29; ++i) std::cout << sumAb[i] << "\t";
  //std::cout << std::endl;
  ATA.fill(0.);
  ATb.fill(0.);
  int k = 0;
  for (int i=0; i<6; ++i) {
    for (int j=i; j<7; ++j) {
      float val = sumAb[k++];
      if (j==6)  {
        ATb(i) = val;
      } else {
        ATA(i,j) = val;
        ATA(j,i) = val;
      }
    }
  }
  count = sumAb[28];
  error = sumAb[27]/count;
  //std::cout << ATA << std::endl << ATb.transpose() << std::endl;
  //std::cout << "\terror&count " << error << " " << count << std::endl;
}

// T_mc: R_model_observation
template<int BLK_SIZE, int D, typename Derived>
__global__ void KernelICPStep(
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    SE3f T_mo, 
    SE3f T_co, 
    const CameraBase<float,D,Derived> cam,
    float dotThr,
    float distThr,
    int N_PER_T,
    Image<float> out
    ) {
  assert(BLK_SIZE >=29);
  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = idx*N_PER_T;
  const int N = pc_m.w_*pc_m.h_;
  const int idE = min(N,(idx+1)*N_PER_T);

  SharedMemory<Vector29fda> smem;
  Vector29fda* sum = smem.getPointer();

  sum[tid] = Vector29fda::Zero();
  for (int id=idS; id<idE; ++id) {
    const int x = id%pc_o.w_;
    const int y = id/pc_o.w_;
    int u, v;
    int res = AssociateModelIntoCurrent<D,Derived>(x, y, pc_m, T_mo,
        T_co, cam, u, v);
    if (res == 0) {
      // found association -> check thresholds;
      Vector3fda n_o_in_m = T_mo.rotation()*n_o(u,v);
      Vector3fda n_mi = n_m(x,y);
      Vector3fda pc_mi = pc_m(x,y);
      Vector3fda pc_oi = pc_o(u,v);
      Vector3fda pc_o_in_m = T_mo * pc_oi;
      const float dot  = n_mi.dot(n_o_in_m);
      const float dist = (pc_mi-pc_o_in_m).norm();
      if (dot > dotThr && dist < distThr && IsValidData(pc_mi)) {
        // association is good -> accumulate
        // if we found a valid association accumulate the A and b for A x = b
        // where x \in se{3} as well as the residual error
        float ab[7];      
        Eigen::Map<Vector3fda> top(&(ab[0]));
        Eigen::Map<Vector3fda> bottom(&(ab[3]));
        // as in mp3guy: 
        top = (pc_o_in_m).cross(n_mi);
        bottom = n_mi;
        ab[6] = n_mi.dot(pc_mi-pc_o_in_m);
        Eigen::Matrix<float,29,1,Eigen::DontAlign> upperTriangle;
        int k=0;
#pragma unroll
        for (int i=0; i<7; ++i) {
          for (int j=i; j<7; ++j) {
            upperTriangle(k++) = ab[i]*ab[j];
          }
        }
        upperTriangle(28) = 1.; // to get number of data points
        sum[tid] += upperTriangle;
      }
    }
  }
  __syncthreads(); //sync the threads
#pragma unroll
  for(int s=(BLK_SIZE)/2; s>1; s>>=1) {
    if(tid < s) {
      sum[tid] += sum[tid+s];
    }
    __syncthreads();
  }
  if(tid < 29) {
    // sum the last two remaining matrixes directly into global memory
    atomicAdd(&out[tid], sum[0](tid)+sum[1](tid));
  }
}

template<int D, typename Derived>
void ICPStep (
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    const SE3f& T_mo, 
    const SE3f& T_cm,
    const CameraBase<float,D,Derived>& cam,
    float dotThr,
    float distThr,
    Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb,
    float& error,
    float& count
    ) {
  const size_t BLK_SIZE = 32;
  size_t N = pc_m.w_*pc_m.h_;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,N/10,BLK_SIZE);
  ManagedDeviceImage<float> out(29,1);
  hipMemset(out.ptr_, 0, 29*sizeof(float));

  KernelICPStep<BLK_SIZE,D,Derived><<<blocks,threads,
    BLK_SIZE*sizeof(Vector29fda)>>>(
        pc_m,n_m,pc_o,n_o,T_mo,T_cm,cam,
        dotThr,distThr,10,out);
  checkCudaErrors(hipDeviceSynchronize());
  ManagedHostImage<float> sumAb(29,1);
  hipMemcpy(sumAb.ptr_,out.ptr_,29*sizeof(float), hipMemcpyDeviceToHost);

  //for (int i=0; i<29; ++i) std::cout << sumAb[i] << "\t";
  //std::cout << std::endl;
  ATA.fill(0.);
  ATb.fill(0.);
  int k = 0;
  for (int i=0; i<6; ++i) {
    for (int j=i; j<7; ++j) {
      float val = sumAb[k++];
      if (j==6)  {
        ATb(i) = val;
      } else {
        ATA(i,j) = val;
        ATA(j,i) = val;
      }
    }
  }
  count = sumAb[28];
  error = sumAb[27]/count;
  //std::cout << ATA << std::endl << ATb.transpose() << std::endl;
  //std::cout << "\terror&count " << error << " " << count << std::endl;
}

// explicit instantiation
template void ICPStep (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o, const SE3f& T_mo, const SE3f& T_cm,
    const CameraBase<float,Camera<float>::NumParams,Camera<float>>& cam,
    float dotThr, float distThr, Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb, float& error, float& count);
template void ICPStep (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o, const SE3f& T_mo, const SE3f& T_cm,
    const CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>>& cam,
    float dotThr, float distThr, Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb, float& error, float& count);

// T_mc: R_model_observation
template<int BLK_SIZE, int D, typename Derived>
__global__ void KernelICPStep(
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector2fda> gradGrey_m,
    Image<float> grey_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    Image<Vector2fda> gradGrey_o,
    Image<float> grey_o,
    SE3f T_mo, 
    SE3f T_co, 
    const CameraBase<float,D,Derived> cam,
    float dotThr,
    float distThr,
    float lambda,
    int N_PER_T,
    Image<float> out
    ) {
  assert(BLK_SIZE >=29);
  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = idx*N_PER_T;
  const int N = pc_m.w_*pc_m.h_;
  const int idE = min(N,(idx+1)*N_PER_T);

  SharedMemory<Vector29fda> smem;
  Vector29fda* sum = smem.getPointer();

  sum[tid] = Vector29fda::Zero();
  for (int id=idS; id<idE; ++id) {
    const int x = id%pc_o.w_;
    const int y = id/pc_o.w_;
    int u, v;
    int res = AssociateModelIntoCurrent<D,Derived>(x, y, pc_m, T_mo,
        T_co, cam, u, v);
    if (res == 0) {
      // found association -> check thresholds;
      Vector3fda n_o_in_m = T_mo.rotation()*n_o(u,v);
      Vector3fda n_mi = n_m(x,y);
      Vector3fda pc_mi = pc_m(x,y);
      Vector3fda pc_oi = pc_o(u,v);
      Vector3fda pc_o_in_m = T_mo * pc_oi;
      const float dot  = n_mi.dot(n_o_in_m);
      const float dist = (pc_mi-pc_o_in_m).norm();
      if (dot > dotThr && dist < distThr && IsValidData(pc_mi)) {
        // association is good -> accumulate
        float I_m = grey_m(x,y);
        Vector2fda gradI_m = gradGrey_m(x,y);
        float I_o = grey_o(u,v); // TODO: maybe interpolate here
        float abI[7];      
        Eigen::Map<Vector6fda> J(&(abI[0]));
        Eigen::Matrix<float,2,3> Jpi = cam.Jproject(pc_o_in_m);
        Eigen::Matrix<float,3,6> Jse3;
        Jse3 << -SO3mat<float>::invVee(pc_o_in_m), Eigen::Matrix3f::Identity();
        J = Jse3.transpose() * Jpi.transpose() * gradI_m;
        abI[6] = -I_m + I_o;
        float ab[7];      
        Eigen::Map<Vector3fda> top(&(ab[0]));
        Eigen::Map<Vector3fda> bottom(&(ab[3]));
        top = (pc_o_in_m).cross(n_mi);
        bottom = n_mi;
        ab[6] = n_mi.dot(pc_mi-pc_o_in_m);
        Eigen::Matrix<float,29,1,Eigen::DontAlign> upperTriangle;
        int k=0;
#pragma unroll
        for (int i=0; i<7; ++i) {
          for (int j=i; j<7; ++j) {
            upperTriangle(k++) = ab[i]*ab[j] + lambda*abI[i]*abI[j];
          }
        }
        upperTriangle(28) = 1.; // to get number of data points
        sum[tid] += upperTriangle;
      }
    }
  }
  __syncthreads(); //sync the threads
#pragma unroll
  for(int s=(BLK_SIZE)/2; s>1; s>>=1) {
    if(tid < s) {
      sum[tid] += sum[tid+s];
    }
    __syncthreads();
  }
  if(tid < 29) {
    // sum the last two remaining matrixes directly into global memory
    atomicAdd(&out[tid], sum[0](tid)+sum[1](tid));
  }
}

template<int D, typename Derived>
void ICPStep (
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector2fda> gradGrey_m,
    Image<float> grey_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    Image<Vector2fda> gradGrey_o,
    Image<float> grey_o,
    const SE3f& T_mo, 
    const SE3f& T_cm,
    const CameraBase<float,D,Derived>& cam,
    float dotThr,
    float distThr,
    float lambda,
    Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb,
    float& error,
    float& count
    ) {
  const size_t BLK_SIZE = 32;
  size_t N = pc_m.w_*pc_m.h_;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,N/10,BLK_SIZE);
  ManagedDeviceImage<float> out(29,1);
  hipMemset(out.ptr_, 0, 29*sizeof(float));

  KernelICPStep<BLK_SIZE,D,Derived><<<blocks,threads,
    BLK_SIZE*sizeof(Vector29fda)>>>(
        pc_m,n_m,gradGrey_m, grey_m, pc_o,n_o, gradGrey_o, grey_o,
        T_mo,T_cm,cam, dotThr,distThr, lambda,10,out);
  checkCudaErrors(hipDeviceSynchronize());
  ManagedHostImage<float> sumAb(29,1);
  hipMemcpy(sumAb.ptr_,out.ptr_,29*sizeof(float), hipMemcpyDeviceToHost);

  //for (int i=0; i<29; ++i) std::cout << sumAb[i] << "\t";
  //std::cout << std::endl;
  ATA.fill(0.);
  ATb.fill(0.);
  int k = 0;
  for (int i=0; i<6; ++i) {
    for (int j=i; j<7; ++j) {
      float val = sumAb[k++];
      if (j==6)  {
        ATb(i) = val;
      } else {
        ATA(i,j) = val;
        ATA(j,i) = val;
      }
    }
  }
  count = sumAb[28];
  error = sumAb[27]/count;
  //std::cout << ATA << std::endl << ATb.transpose() << std::endl;
  //std::cout << "\terror&count " << error << " " << count << std::endl;
}

// explicit instantiation
template void ICPStep (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, 
    Image<Vector2fda> gradGrey_m, Image<float> grey_m,
    Image<Vector3fda> pc_o, Image<Vector3fda> n_o, 
    Image<Vector2fda> gradGrey_o, Image<float> grey_o,
    const SE3f& T_mo, const SE3f& T_cm,
    const CameraBase<float,Camera<float>::NumParams,Camera<float>>& cam,
    float dotThr, float distThr, float lambda,
    Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb, float& error,
    float& count);
template void ICPStep (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, 
    Image<Vector2fda> gradGrey_m, Image<float> grey_m,
    Image<Vector3fda> pc_o, Image<Vector3fda> n_o, 
    Image<Vector2fda> gradGrey_o, Image<float> grey_o,
    const SE3f& T_mo, const SE3f& T_cm,
    const CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>>& cam,
    float dotThr, float distThr, float lambda,
    Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb, float& error,
    float& count);

// T_mc: R_model_observation
template<int BLK_SIZE, int D, typename Derived>
__global__ void KernelICPStep(
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> g_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    Image<Vector3fda> g_o,
    SE3f T_mo, 
    SE3f T_co, 
    const CameraBase<float,D,Derived> cam,
    float dotThr,
    float distThr,
    int N_PER_T,
    Image<float> out
    ) {
  assert(BLK_SIZE >=29);
  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = idx*N_PER_T;
  const int N = pc_m.w_*pc_m.h_;
  const int idE = min(N,(idx+1)*N_PER_T);

  SharedMemory<Vector29fda> smem;
  Vector29fda* sum = smem.getPointer();

  sum[tid] = Vector29fda::Zero();
  for (int id=idS; id<idE; ++id) {
    const int x = id%pc_o.w_;
    const int y = id/pc_o.w_;
    int u, v;
    int res = AssociateModelIntoCurrent<D,Derived>(x, y, pc_m, T_mo,
        T_co, cam, u, v);
    if (res == 0) {
      // found association -> check thresholds;
      Vector3fda n_o_in_m = T_mo.rotation()*n_o(u,v);
      Vector3fda n_mi = n_m(x,y);
      Vector3fda g_mi = g_m(x,y).normalized();
      Vector3fda pc_mi = pc_m(x,y);
      Vector3fda pc_oi = pc_o(u,v);
      Vector3fda pc_o_in_m = T_mo * pc_oi;
      const float dot  = n_mi.dot(n_o_in_m);
      const float dist = (pc_mi-pc_o_in_m).norm();
      if (dot > dotThr && dist < distThr && IsValidData(pc_mi)) {
        // association is good -> accumulate
        // if we found a valid association accumulate the A and b for A x = b
        // where x \in se{3} as well as the residual error
        
        // contribution by surface normal
        float ab[7];      
        Eigen::Map<Vector3fda> top(&(ab[0]));
        Eigen::Map<Vector3fda> bottom(&(ab[3]));
        // as in mp3guy: 
        top = (pc_o_in_m).cross(n_mi);
        bottom = n_mi;
        ab[6] = n_mi.dot(pc_mi-pc_o_in_m);

        // contribution by 3D gradients
        float abg[7];      
        Eigen::Map<Vector3fda> topg(&(abg[0]));
        Eigen::Map<Vector3fda> bottomg(&(abg[3]));
        if (IsValidData(g_mi)) {
            topg = (pc_o_in_m).cross(g_mi);
            bottomg = g_mi;
            abg[6] = g_mi.dot(pc_mi-pc_o_in_m);
        } else {
          topg = Vector3fda::Zero(); 
          bottomg = Vector3fda::Zero(); 
          abg[6] = 0.;
        }

        Eigen::Matrix<float,29,1,Eigen::DontAlign> upperTriangle;
        int k=0;
#pragma unroll
        for (int i=0; i<7; ++i) {
          for (int j=i; j<7; ++j) {
            upperTriangle(k++) = ab[i]*ab[j] + abg[i]*abg[j];
          }
        }
        upperTriangle(28) = 1.; // to get number of data points
        sum[tid] += upperTriangle;
      }
    }
  }
  __syncthreads(); //sync the threads
#pragma unroll
  for(int s=(BLK_SIZE)/2; s>1; s>>=1) {
    if(tid < s) {
      sum[tid] += sum[tid+s];
    }
    __syncthreads();
  }
  if(tid < 29) {
    // sum the last two remaining matrixes directly into global memory
    atomicAdd(&out[tid], sum[0](tid)+sum[1](tid));
  }
}

template<int D, typename Derived>
void ICPStep (
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> g_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    Image<Vector3fda> g_o,
    const SE3f& T_mo, 
    const SE3f& T_cm,
    const CameraBase<float,D,Derived>& cam,
    float dotThr,
    float distThr,
    Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb,
    float& error,
    float& count
    ) {
  const size_t BLK_SIZE = 32;
  size_t N = pc_m.w_*pc_m.h_;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,N/10,BLK_SIZE);
  ManagedDeviceImage<float> out(29,1);
  hipMemset(out.ptr_, 0, 29*sizeof(float));

  KernelICPStep<BLK_SIZE,D,Derived><<<blocks,threads,
    BLK_SIZE*sizeof(Vector29fda)>>>(
        pc_m,n_m,g_m,pc_o,n_o,g_o,T_mo,T_cm,cam,
        dotThr,distThr,10,out);
  checkCudaErrors(hipDeviceSynchronize());
  ManagedHostImage<float> sumAb(29,1);
  hipMemcpy(sumAb.ptr_,out.ptr_,29*sizeof(float), hipMemcpyDeviceToHost);

  //for (int i=0; i<29; ++i) std::cout << sumAb[i] << "\t";
  //std::cout << std::endl;
  ATA.fill(0.);
  ATb.fill(0.);
  int k = 0;
  for (int i=0; i<6; ++i) {
    for (int j=i; j<7; ++j) {
      float val = sumAb[k++];
      if (j==6)  {
        ATb(i) = val;
      } else {
        ATA(i,j) = val;
        ATA(j,i) = val;
      }
    }
  }
  count = sumAb[28];
  error = sumAb[27]/count;
  //std::cout << ATA << std::endl << ATb.transpose() << std::endl;
  //std::cout << "\terror&count " << error << " " << count << std::endl;
}

// explicit instantiation
template void ICPStep (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> g_m, 
    Image<Vector3fda> pc_o, Image<Vector3fda> n_o, Image<Vector3fda> g_o,
    const SE3f& T_mo, const SE3f& T_cm,
    const CameraBase<float,Camera<float>::NumParams,Camera<float>>& cam,
    float dotThr, float distThr, Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb, float& error, float& count);
template void ICPStep (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> g_m, 
    Image<Vector3fda> pc_o, Image<Vector3fda> n_o, Image<Vector3fda> g_o,
    const SE3f& T_mo, const SE3f& T_cm,
    const CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>>& cam,
    float dotThr, float distThr, Eigen::Matrix<float,6,6,Eigen::DontAlign>& ATA,
    Eigen::Matrix<float,6,1,Eigen::DontAlign>& ATb, float& error, float& count);

// T_mc: R_model_observation
template<int BLK_SIZE, int D, class Derived>
__global__ void KernelICPVisualizeAssoc(
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    SE3f T_mo,
    const CameraBase<float,D,Derived> cam,
    float dotThr,
    float distThr,
    int N_PER_T,
    Image<float> assoc_m,
    Image<float> assoc_o
    ) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idS = idx*N_PER_T;
  const int N = pc_m.w_*pc_m.h_;
  const int idE = min(N,(idx+1)*N_PER_T);

  for (int id=idS; id<idE; ++id) {
    const int x = id%pc_m.w_;
    const int y = id/pc_m.w_;
    int u, v;
    int res = AssociateModelIntoCurrent<D,Derived>(x, y, pc_m, T_mo,
        tdp::SE3f(), cam, u, v);
    if (res == 0) {
      // found association -> check thresholds;
      Vector3fda pc_mi = pc_m(x,y);
      Vector3fda n_mi = n_m(x,y);
      Vector3fda n_o_in_m = T_mo.rotation() * n_o(u,v);
      Vector3fda pc_oi = pc_o(u,v);
      Vector3fda pc_o_in_m = T_mo * pc_oi;
      float dot  = n_mi.dot(n_o_in_m);
      float dist = (pc_mi-pc_o_in_m).norm();
      if (dot > dotThr && dist < distThr && IsValidData(pc_mi)) {
        // association is good -> accumulate
        //assoc_m(u,v) = n_mi.dot(-pc_mi+pc_o_in_m);
        //assoc_o(x,y) = n_mi.dot(-pc_mi+pc_o_in_m);
        assoc_m(x,y) = n_mi.dot(-pc_mi+pc_o_in_m);
        //        assoc_o(u,v) = n_mi.dot(-pc_mi+pc_o_in_m);
        //          if (threadIdx.x < 3) printf("%d,%d and %d,%d\n", x,y,u,v);
      }
    } else if (res < 3) {
      assoc_o(x,y) = res;
    }
  }
}

template<int D, typename Derived>
void ICPVisualizeAssoc (
    Image<Vector3fda> pc_m,
    Image<Vector3fda> n_m,
    Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o,
    const SE3f& T_mo,
    const CameraBase<float,D,Derived>& cam,
    float angleThr,
    float distThr,
    Image<float>& assoc_m,
    Image<float>& assoc_o
    ) {
  size_t N = pc_m.w_*pc_m.h_;
  dim3 threads, blocks;
  ComputeKernelParamsForArray(blocks,threads,N/10,256);
  KernelICPVisualizeAssoc<256,D,Derived><<<blocks,threads>>>(pc_m,n_m,pc_o,n_o,
      T_mo,cam, cos(angleThr*M_PI/180.),distThr,10,assoc_m, assoc_o);
  checkCudaErrors(hipDeviceSynchronize());
}

template void ICPVisualizeAssoc (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o, const SE3f& T_mo,
    const CameraBase<float,Camera<float>::NumParams,Camera<float>>& cam,
    float angleThr, float distThr, Image<float>& assoc_m, Image<float>& assoc_o);
template void ICPVisualizeAssoc (
    Image<Vector3fda> pc_m, Image<Vector3fda> n_m, Image<Vector3fda> pc_o,
    Image<Vector3fda> n_o, const SE3f& T_mo,
    const CameraBase<float,CameraPoly3<float>::NumParams,CameraPoly3<float>>& cam,
    float angleThr, float distThr, Image<float>& assoc_m, Image<float>& assoc_o);


}
