#include "hip/hip_runtime.h"


#include <math.h>
#include <iostream>
#include <Eigen/Core>
#include <Eigen/Dense>
#include <tdp/cuda/hip/hip_runtime.h>
#include <tdp/tsdf/tsdf_projective.h>
#include <tdp/camera/projective_math.h>
#include <tdp/camera/camera.h>

namespace tdp {

__global__
void KernelRayTraceProjectiveTSDF(Volume<float> tsdf, Image<float> d, 
    SE3<float> T_rd, Camera<float> camR, Camera<float> camD,
    float rho0, float drho, float mu) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if (idx < d.w_ && idy < d.h_) {
    d(idx,idy) = NAN;
    Eigen::Vector3f n(0,0,-1);
    Eigen::Vector2f u_d(idx,idy);
    // iterate over depth starting from sensor; detect 0 crossing
    float tsdfValPrev = -1.01;
    for (size_t id=tsdf.d_; id>0; --id) {
      float rho = rho0 + drho*(id-1);  // invers depth
      Eigen::Vector3f nd = n*rho;
      Eigen::Matrix3f H = (T_rd.rotation().matrix()-T_rd.translation()*nd.transpose());
      Eigen::Vector2f u_r = camR.Project(H*camD.Unproject(u_d(0), u_d(1), 1.));
      int x = floor(u_r(0)+0.5);
      int y = floor(u_r(1)+0.5);
      if (0<=x&&x<tsdf.w_ && 0<=y&&y<tsdf.h_) {
        float tsdfVal = tsdf(x,y,id-1);
        if (tsdfVal <= 0. && tsdfValPrev >= 0.) {
          // detected 0 crossing -> interpolate
          float idf = id+tsdfValPrev/(tsdfVal-tsdfValPrev);
          d(idx,idy) = 1./(rho0 + drho*idf);
          break;
        }
        tsdfValPrev = tsdfVal;
      }
    }
  }
}


__global__ 
void KernelAddToProjectiveTSDF(Volume<float> tsdf, Volume<float> W,
    Image<float> d, SE3<float> T_rd, SE3<float> T_dr, Camera<float>
    camR, Camera<float>camD, float rho0, float drho, float mu) {
  // kernel over all pixel locations and depth locations in the TSDF
  // volume
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  const int idy = threadIdx.y + blockDim.y * blockIdx.y;
  const int idz = threadIdx.z + blockDim.z * blockIdx.z;

  if (idx < tsdf.w_ && idy < tsdf.h_ && idz < tsdf.d_) {
    // unproject point in reference frame
    float rho_r = rho0+drho*idz;
    Eigen::Vector3f p_r = camR.Unproject(idx,idy,1./rho_r);
    // project the point into the depth frame
    Eigen::Vector2f u_d = camD.Project(T_dr*p_r);
    int x = floor(u_d(0)+0.5);
    int y = floor(u_d(1)+0.5);
    if (0<=x&&x<d.w_ && 0<=y&&y<d.h_) {
      const float z_d = d(x, y);
      const float lambda = camD.Unproject(u_d(0),u_d(1),1.).norm();
      const float z_tsdf = (T_rd.translation()-p_r).norm()/lambda;
      const float eta = z_d - z_tsdf;
      if (eta >= -mu) {
        const float etaOverMu = eta/mu;
        const float psi = (etaOverMu>1.f?1.f:etaOverMu);
        // TODO can use other weights as well (like incidence angle)
        const float Wnew = 1.;
        tsdf(idx,idy,idz) = (W(idx,idy,idz)*tsdf(idx,idy,idz) 
            + Wnew*psi)/(W(idx,idy,idz)+Wnew);
        W(idx,idy,idz) = min(W(idx,idy,idz)+Wnew, 100.f);
      }
    }
  }
}

void AddToProjectiveTSDF(Volume<float> tsdf, Volume<float> W, Image<float> d, 
    SE3<float> T_rd, Camera<float> camR, Camera<float>camD,
    float rho0, float drho, float mu) {
  dim3 threads, blocks;
  ComputeKernelParamsForVolume(blocks,threads,tsdf,16,16,4);
  KernelAddToProjectiveTSDF<<<blocks,threads>>>(tsdf, W, d, T_rd, T_rd.Inverse(), camR, camD, rho0, drho, mu);
  checkCudaErrors(hipDeviceSynchronize());
}

void RayTraceProjectiveTSDF(Volume<float> tsdf, Image<float> d, 
    SE3<float> T_rd, Camera<float> camR, Camera<float>camD,
    float rho0, float drho, float mu) {
  dim3 threads, blocks;
  ComputeKernelParamsForImage(blocks,threads,d,32,32);
  KernelRayTraceProjectiveTSDF<<<blocks,threads>>>(tsdf, d, T_rd, camR, camD, rho0, drho, mu);
  checkCudaErrors(hipDeviceSynchronize());
}

}
